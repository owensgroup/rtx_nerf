#include "hip/hip_runtime.h"
#include <iostream>
#include <cmath>
#include <string>
#include <vector>
#include <cstdio>
#include <fstream>
#include <math.h>
#include <thrust/device_vector.h>
#include <thrust/device_ptr.h>
#include "stdio.h"
#include "sampler.h"

#include "optix_function_table_definition.h"
#include "optix_stubs.h"
#include "optix.h"
#include "optix_types.h"

#define STB_IMAGE_IMPLEMENTATION
#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "stb_image.h"
#include "stb_image_write.h"
#include "tiny-cuda-nn/common.h"
#include "tiny-cuda-nn/gpu_matrix.h"
#include "tiny-cuda-nn/config.h"
#include "tiny-cuda-nn/reduce_sum.h"
#include <json/json.h>
#include "rtx/include/params.h"
#include "rtx/include/rtxFunctions.h"

#include "data_loader.h"
#include "vol_render.h"
// Configure the model
nlohmann::json config = {
	{"loss", {
		{"otype", "L2"}
	}},
    // adam optimizer decays from 5e-4 to 5e-5
	{"optimizer", {
		{"otype", "Adam"},
		{"learning_rate", 1e-3},
        {"beta1", 0.9},
        {"beta2", 0.999},
        {"epsilon", 1e-8}
	}},
	{"encoding", {
        {"otype", "Composite"},
        {"nested", {
            {
                {"n_dims_to_encode", 3}, // Spatial dims
                {"otype", "Frequency"},
                {"n_frequencies", 10}
            },
            {
                {"n_dims_to_encode", 2}, // Non-linear appearance dims.
                {"otype", "Frequency"},
                {"n_bins", 4}
            }
        }}
    }},
	{"network", {
		{"otype", "FullyFusedMLP"},
		{"activation", "ReLU"},
		{"output_activation", "Sigmoid"},
		{"n_neurons", 128},
		{"n_hidden_layers", 8}
	}}
};

template<typename T>
void printGPUMatrix(
    const tcnn::GPUMatrix<T>& matrix,
    int n_rows, int n_cols) {
    // Get the dimensions of the matrix
    uint32_t rows = matrix.rows();
    uint32_t cols = matrix.cols();

    // Allocate host memory to store the matrix data
    T* hostData = new T[rows * cols];

    // Copy the matrix data from GPU to host
    hipMemcpy(hostData, matrix.data(), sizeof(T) * rows * cols, hipMemcpyDeviceToHost);

    // Print the matrix values
    for (uint32_t i = 0; i < n_rows; i++) {
        for (uint32_t j = 0; j < n_cols; j++) {
            std::cout << hostData[i * cols + j] << " ";
        }
        std::cout << std::endl;
    }

    // Free the host memory
    delete[] hostData;
}

__global__ void printFloats(float* gpuPointer, int size)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    if (tid < size)
    {
        printf("Value at index %d: %f\n", tid, gpuPointer[tid]);
    }
}
__global__ void print_batch(float* batch, int batch_size, int image_size) {
    //printf("HELLO???\n");
    // int tid = blockIdx.x * blockDim.x + threadIdx.x;
    // if (tid < batch_size) {
    //     printf("Batch %d\n", tid);
    //     for (int i = 0; i < image_size; ++i) {
    //         printf("%f ", batch[tid * image_size + i]);
    //     }
    //     printf("\n");
    // }
}

__global__ void gatherIntersections(
    float3* d_start_points, 
    float3* d_end_points, 
    int* d_num_hits, 
    float3* d_intersect_start,
    float3* d_intersect_end,
    int width, int height, int grid_size)
{
    // Calculate the index of the pixel this thread should process.
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < width && y < height)
    {
        // Calculate the base index for this pixel in the d_start_points and d_end_points arrays.
        int base_index = (y * width + x) * grid_size;

        // Find the number of grid cells hit by the ray from this pixel.
        int num_hits = d_num_hits[y * width + x];

        // For each hit, gather the entry and exit points.
        for (int i = 0; i < num_hits; ++i)
        {
            float3 start_point = d_start_points[base_index + i];
            float3 end_point = d_end_points[base_index + i];

            // Store the intersection points.
            d_intersect_start[2 * (base_index + i)] = start_point;
            d_intersect_end[2 * (base_index + i)] = end_point;
        }
    }
}
 
// Creates a grid of Axis-aligned bounding boxes with specified resolution
// Bounding box coordinates are specified in normalized coordinates from -1 to 1
// TODO: make this a CUDA kernel
std::vector<OptixAabb> make_grid(int resolution) {
    std::vector<OptixAabb> grid;
    float box_length = 2.0f/ (float)resolution;
    for(int x = 0; x < resolution; x++) {
        for(int y = 0; y < resolution; y++) {
            for(int z = 0; z < resolution; z++) {
                OptixAabb aabb;
                aabb.minX = -1.0f + (float)x * box_length;
                aabb.maxX = -1.0f + x * box_length + box_length;
                aabb.minY = -1.0f + y * box_length;
                aabb.maxY = -1.0f + y * box_length + box_length;
                aabb.minZ = -1.0f + z * box_length;
                aabb.maxZ = -1.0f + z * box_length + box_length;
                grid.push_back(aabb);
                //std::printf("aabb (%.2f %.2f %.2f) (%.2f %.2f %.2f)\n",
                //        aabb.minX, aabb.minY, aabb.minZ, aabb.maxX, aabb.maxY, aabb.maxZ);
            }
        }
    }
    return grid;
}

void printGPUMem() {
    size_t freeMem, totalMem;
    hipMemGetInfo(&freeMem, &totalMem);
    size_t usedMem = totalMem - freeMem;
    std::cout << "GPU Memory Usage: " << usedMem / 1024 / 1024 << " MB" << std::endl;
}

//auto model = tcnn::create_from_config(n_input_dims, n_output_dims, config);

#define EPOCHS 10
#define BATCH_SIZE tcnn::BATCH_SIZE_GRANULARITY*160
#define DATASET_SIZE 1000

RTXDataHolder *rtx_dataholder;

__global__ void print_intersections(float3* start, float3* end, int* num_hits, int num_prim) {
    printf("Intersections\n");
    for (int i = 0; i < 100; ++i) {
        printf("ray (%i): %i hits\n", i, num_hits[i]); // origin = (%.2f, %.2f, %.2f)\n  ",
        for (int j = 0; j < num_hits[i]; ++j) {
            float3 s = start[i*num_prim + j];
            float3 e = end[i*num_prim + j];
            printf("   (%.2f %.2f %.2f) (%.2f %.2f %.2f)\n", s.x, s.y, s.z, e.x, e.y, e.z);
        }
    }
}

__global__ void convertHalfToFloat(__half* input, float* output, int size) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < size) {
        output[tid] = __half2float(input[tid]);
    }
}

__global__ void floatToHalf(float* input, __half* output, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        output[idx] = __float2half(input[idx]);
    }
}

__global__ void print_int_arr(int* arr, int size) {
    // Print the first 10 and last 10 elements in the buffer
    printf("First 10 elements:\n");
    for (int i = 0; i < 10; ++i) {
        printf("%d ", arr[i]);
    }
    printf("\n");

    printf("Last 10 elements:\n");
    for (int i = size - 10; i < size; ++i) {
        printf("%d ", arr[i]);
    }
    printf("\n");
}



__global__ void print_float_arr(float* arr, int size) {
    printf("Printing float array\n");
    for (int i = 0; i < 10; ++i) {
        printf("%f ", arr[i]);
    }
    printf("\n");
}

__global__ void print_float2_arr(float2* arr, int width, int height) {
    printf("Printing float2 array\n");
    for (int i = 0; i < 10; ++i) {
        for(int j = 0; j < 10; ++j) {
            printf("%f %f ", arr[i * width + j].x, arr[i * width + j].y);
        }
        printf("\n");
    }
    printf("\n");
}

__global__ void print_float5_arr(float* arr, int size) {
    printf("Printing first 32 points \n");
    for(int i = 0; i < 64; ++i) {
        printf("%f %f %f %f %f\n", arr[i*5], arr[i*5+1], arr[i*5+2], arr[i*5+3], arr[i*5+4]);
    }
    printf("\n");
    printf("Printing last 32 points \n");
    for(int i = size-64; i < size; ++i) {
        printf("%f %f %f %f %f\n", arr[i*5], arr[i*5+1], arr[i*5+2], arr[i*5+3], arr[i*5+4]);
    }
    printf("\n");
}

__global__ void print_float3_arr(float3* arr, int size) {
    printf("Printing float3 array\n");
    printf("Printing first 32 points \n");
    for(int i = 0; i < 32; ++i) {
        printf("%f %f %f\n", arr[i].x, arr[i].y, arr[i].z);
    }
    printf("\n");
    printf("Printing last 32 points \n");
    for(int i = size-32; i < size; ++i) {
        printf("%f %f %f\n", arr[i].x, arr[i].y, arr[i].z);
    }
    printf("\n");
}

__global__ void print_float4_arr(float* arr, int size) {
    printf("Printing float4 array\n");
    printf("Printing first 32 points \n");
    for(int i = 0; i < 32; ++i) {
        printf("%f %f %f %f\n", arr[i*4], arr[i*4+1], arr[i*4+2], arr[i*4+3]);
    }
    printf("\n");
    printf("Printing last 32 points \n");
    for(int i = size-32; i < size; ++i) {
        printf("%f %f %f %f\n", arr[i*4], arr[i*4+1], arr[i*4+2], arr[i*4+3]);
    }
    printf("\n");
}

struct RayPayload {
    int num_hits;
    float3 origin;
    float2 view_dir;
    float* t_start;
    float* t_end;
    float3* start_points;
    float3* end_points;
    float3 pixel_color_gt;
};

int main() {
    // load data from files
    // TODO: take images and poses from json and load into DataLoader
    int n_input_dims = 5;
    int n_output_dims = 4;
    int batch_size = BATCH_SIZE;
    auto model = tcnn::create_from_config(n_input_dims, n_output_dims, config);
    model.optimizer->allocate(model.network);
    int num_epochs = EPOCHS;
    std::cout << "---------------------- Loading Data ----------------------\n";
    // Loads the Training, validation, and test sets from the synthetic lego scene
    std::vector<ImageDataset> datasets = load_data(SceneType::SYNTHETIC, SyntheticName::LEGO);
    auto train_set = datasets[0];
    unsigned int width = train_set.image_width;
    unsigned int height = train_set.image_height;
    unsigned int channels = train_set.image_channels;
    float training_focal = train_set.focal;
    float aspect_ratio = (float)width / (float)height;
    float focal_length = 1.0f / tan(0.5f * training_focal);
    size_t image_size = width * height * channels;
    // get training dataset from datasets
    std::vector<float*> training_images = datasets[0].images;
    std::vector<float*> training_poses = datasets[0].poses;
    std::cout << "---------------------- Data Loaded ----------------------\n\n\n";
    // Initialize our Optix Program Groups and Pipeline
    // We also build our initial dense acceleration structure of AABBs

    std::cout << "---------------------- Initializing Optix ----------------------\n";
    hipStream_t inference_stream;
    hipStream_t training_stream;
    CUDA_CHECK(hipStreamCreate(&inference_stream));
    CUDA_CHECK(hipStreamCreate(&training_stream));
    std::string ptx_filename = BUILD_DIR "bin/ptx/optixPrograms.ptx";

    rtx_dataholder = new RTXDataHolder();
    std::cout << "Initializing Context \n";
    rtx_dataholder->initContext();
    std::cout << "Reading PTX file and creating modules \n";
    rtx_dataholder->createModule(ptx_filename);
    std::cout << "Creating Optix Program Groups \n";
    rtx_dataholder->createProgramGroups();
    std::cout << "Linking Pipeline \n";
    rtx_dataholder->linkPipeline(false);
    std::cout << "Building Shader Binding Table (SBT) \n";
    rtx_dataholder->buildSBT();
    
    // Build our initial dense acceleration structure
    int grid_resolution = 8;
    std::cout << "Building Acceleration Structure \n";
    std::vector<OptixAabb> grid = make_grid(grid_resolution);
    int num_primitives = grid.size();
    
    OptixAabb* d_aabb = rtx_dataholder->initAccelerationStructure(grid);
    std::cout << "Done Building Acceleration Structure \n";
    std::cout << "---------------------- Done Initializing Optix ----------------------\n\n\n";

    std::cout << "Allocating Buffers on GPU" << std::endl;
    float* d_image, *d_look_at;
    CUDA_CHECK(hipMalloc((void **)&d_image, image_size * sizeof(float)));
    CUDA_CHECK(hipMalloc((void **)&d_look_at, 16 * sizeof(float)));
    std::cout << "Image Buffers Allocated on GPU" << std::endl;

    // first generate rays for each pixel

    // Allocate buffers to hold outputs from ray intersection tests
    // start and end points are equal to # of AABBs in AS per ray [width * height * num_primitives]
    float3 *d_start_points;
    float3 *d_end_points;
    float3 *d_ray_origins;
    int *d_num_hits;
    float2 *d_view_dir;
    float* d_pixels;
    float* d_temp_out;
    float* d_t_start;
    float* d_t_end;
    tcnn::network_precision_t* d_pixels_half;

    CUDA_CHECK(hipMalloc((void**)&d_ray_origins, width * height * sizeof(float3)));
    CUDA_CHECK(hipMalloc((void**)&d_t_start, width * height * 3 * grid_resolution * sizeof(float)));
    CUDA_CHECK(hipMalloc((void**)&d_t_end, width * height * 3 * grid_resolution * sizeof(float)));
    CUDA_CHECK(hipMalloc((void **)&d_start_points, width * height * 3 * grid_resolution * sizeof(float3)));
    CUDA_CHECK(hipMalloc((void **)&d_end_points, width * height * 3 * grid_resolution * sizeof(float3)));
    CUDA_CHECK(hipMalloc((void **)&d_num_hits, width * height * sizeof(int)));
    CUDA_CHECK(hipMalloc((void **)&d_view_dir, width * height * sizeof(float2)));
    std::cout << "Ray Intersection Buffers Allocated on GPU" << std::endl;

    CUDA_CHECK(hipMalloc((void**)&d_pixels, batch_size * sizeof(float) * 3));
    CUDA_CHECK(hipMalloc((void **)&d_temp_out, batch_size * n_output_dims * sizeof(float)));
    CUDA_CHECK(hipMalloc((void **)&d_pixels_half, batch_size * sizeof(tcnn::network_precision_t) * 3));
    

    

    Params *d_param;
    CUDA_CHECK(hipMalloc((void **)&d_param, sizeof(Params)));
    std::cout << "Params Buffer Allocated on GPU" << std::endl;

    float3* h_origin;
    float2* h_view_dir;
    int* h_num_hits;
    float* h_t_start;
    float* h_t_end;
    float3* h_start_points;
    float3* h_end_points;

    h_origin = (float3*)malloc(width * height * sizeof(float3));
    h_view_dir = (float2*)malloc(width * height * sizeof(float2));
    h_num_hits = (int*)malloc(width * height * sizeof(int));
    h_t_start = (float*)malloc(width * height * 3 * grid_resolution * sizeof(float));
    h_t_end = (float*)malloc(width * height * 3 * grid_resolution * sizeof(float));
    h_start_points = (float3*)malloc(width * height * 3 * grid_resolution * sizeof(float3));
    h_end_points = (float3*)malloc(width * height * 3 * grid_resolution * sizeof(float3));
    std::vector<RayPayload> ray_payloads;
    // Loop through training data and build dataset
    // dataset consists of ray_payloads and ground truth pixel colors
    // ray_payloads: (origin, dir, num_hits, t_start, t_end)
    for(int i = 0; i < training_images.size(); i++) {
        float* image = training_images[i];
        float* look_at = training_poses[i];
        // transfer image and look_at to GPU
        

        CUDA_CHECK(hipMemcpyAsync(d_image, image, image_size * sizeof(float), hipMemcpyHostToDevice, inference_stream));
        CUDA_CHECK(hipMemcpyAsync(d_look_at, look_at, 16 * sizeof(float), hipMemcpyHostToDevice, inference_stream));

        // Memset ray intersection buffers
        CUDA_CHECK(hipMemsetAsync(d_start_points, -2, width * height * 3 * grid_resolution * sizeof(float3)));
        CUDA_CHECK(hipMemsetAsync(d_end_points, -2, width * height * 3 * grid_resolution * sizeof(float3)));
        CUDA_CHECK(hipMemsetAsync(d_t_start, -2, width * height * 3 * grid_resolution * sizeof(float)));
        CUDA_CHECK(hipMemsetAsync(d_t_end, -2, width * height * 3 * grid_resolution * sizeof(float)));
        CUDA_CHECK(hipMemsetAsync(d_view_dir, -2, width * height * sizeof(float2)));
        CUDA_CHECK(hipMemsetAsync(d_ray_origins, -2, width * height * sizeof(float3))); 
        CUDA_CHECK(hipMemsetAsync(d_num_hits, 0, width * height * sizeof(int)));

        Params params;
        float d =  2.0f / grid_resolution;
        params.delta = make_float3(d, d, d);
        params.min_point = make_float3(-1, -1, -1);
        params.max_point = make_float3(1, 1, 1);
        params.intersection_arr_size = 3 * grid_resolution;
        params.width = width;
        params.height = height;
        params.focal_length = focal_length;
        params.aspect_ratio = aspect_ratio;
        params.handle = rtx_dataholder->gas_handle;
        params.aabb = d_aabb;
        params.start_points = d_start_points;
        params.end_points = d_end_points;
        params.t_start = d_t_start;
        params.t_end = d_t_end;
        params.num_hits = d_num_hits;
        params.num_primitives = num_primitives;
        params.look_at = d_look_at;
        params.viewing_direction = d_view_dir;
        params.ray_origins = d_ray_origins;

        CUDA_CHECK(hipMemcpy(d_param, &params, sizeof(params), hipMemcpyHostToDevice));
        const OptixShaderBindingTable &sbt_ray_march = rtx_dataholder->sbt_ray_march;
        std::cout << "Launching Ray Tracer in Ray Marching Mode (" << width*height << " rays)\n";
        OPTIX_CHECK(optixLaunch(rtx_dataholder->pipeline_ray_march, inference_stream,
                                reinterpret_cast<hipDeviceptr_t>(d_param),
                                sizeof(Params), &sbt_ray_march, width, height, 1));
        CUDA_CHECK(hipStreamSynchronize(inference_stream));
        d_start_points = params.start_points;
        d_end_points = params.end_points;
        d_t_start = params.t_start;
        d_t_end = params.t_end;
        d_num_hits = params.num_hits;
        d_ray_origins = params.ray_origins;

        CUDA_CHECK(hipMemcpy(h_origin, d_ray_origins, width * height * sizeof(float3), hipMemcpyDeviceToHost));
        CUDA_CHECK(hipMemcpy(h_view_dir, d_view_dir, width * height * sizeof(float2), hipMemcpyDeviceToHost));
        CUDA_CHECK(hipMemcpy(h_num_hits, d_num_hits, width * height * sizeof(int), hipMemcpyDeviceToHost));
        CUDA_CHECK(hipMemcpy(h_t_start, d_t_start, width * height * 3 * grid_resolution * sizeof(float), hipMemcpyDeviceToHost));
        CUDA_CHECK(hipMemcpy(h_t_end, d_t_end, width * height * 3 * grid_resolution * sizeof(float), hipMemcpyDeviceToHost));
        CUDA_CHECK(hipMemcpy(h_start_points, d_start_points, width * height * 3 * grid_resolution * sizeof(float3), hipMemcpyDeviceToHost));
        CUDA_CHECK(hipMemcpy(h_end_points, d_end_points, width * height * 3 * grid_resolution * sizeof(float3), hipMemcpyDeviceToHost));

        for(int i = 0; i < width * height; i++) {
            RayPayload payload;
            payload.origin = h_origin[i];
            payload.view_dir = h_view_dir[i];
            payload.num_hits = h_num_hits[i];
            payload.t_start = (float*)malloc(h_num_hits[i] * sizeof(float));
            payload.t_end = (float*)malloc(h_num_hits[i] * sizeof(float));
            payload.start_points = (float3*)malloc(h_num_hits[i] * sizeof(float3));
            payload.end_points = (float3*)malloc(h_num_hits[i] * sizeof(float3));
            for(int j = 0; j < payload.num_hits; j++) {
                payload.t_start[j] = h_t_start[i * 3 * grid_resolution + j];
                payload.t_end[j] = h_t_end[i * 3 * grid_resolution + j];
                payload.start_points[j] = h_start_points[i * 3 * grid_resolution + j];
                payload.end_points[j] = h_end_points[i * 3 * grid_resolution + j];
            }
            payload.pixel_color_gt = make_float3(image[i * 3], image[i * 3 + 1], image[i * 3 + 2]);
            ray_payloads.push_back(payload);
        }
    }
    free(h_origin);
    free(h_view_dir);
    free(h_num_hits);
    free(h_t_start);
    free(h_t_end);
    free(h_start_points);
    free(h_end_points);

    hipFree(d_ray_origins);
    hipFree(d_t_start);
    hipFree(d_t_end);
    hipFree(d_start_points);
    hipFree(d_end_points);
    hipFree(d_num_hits);
    hipFree(d_view_dir);
    std::cout << "---------------------- Done Generating Rays ----------------------\n\n\n";

    // Print 10 random payloads from ray_payloads
    // std::cout << "Random Ray Payloads:" << std::endl;
    // for (int i = 0; i < 10; i++) {
    //     int random_index = rand() % ray_payloads.size();
    //     RayPayload random_payload = ray_payloads[random_index];
    //     std::cout << "Payload " << i+1 << ":" << std::endl;
    //     std::cout << "Origin: (" << random_payload.origin.x << ", " << random_payload.origin.y << ", " << random_payload.origin.z << ")" << std::endl;
    //     std::cout << "View Direction: (" << random_payload.view_dir.x << ", " << random_payload.view_dir.y << ")" << std::endl;
    //     std::cout << "Number of Hits: " << random_payload.num_hits << std::endl;
    //     std::cout << "T Start: ";
    //     for (int j = 0; j < random_payload.num_hits; j++) {
    //         std::cout << random_payload.t_start[j] << " ";
    //     }
    //     std::cout << std::endl;
    //     std::cout << "T End: ";
    //     for (int j = 0; j < random_payload.num_hits; j++) {
    //         std::cout << random_payload.t_end[j] << " ";
    //     }

    //     // print start and end points
    //     std::cout << std::endl;
    //     std::cout << "Start Points: ";
    //     for (int j = 0; j < random_payload.num_hits; j++) {
    //         std::cout << "(" << random_payload.start_points[j].x << ", " << random_payload.start_points[j].y << ", " << random_payload.start_points[j].z << ") ";
    //     }
    //     std::cout << std::endl;
    //     std::cout << "End Points: ";
    //     for (int j = 0; j < random_payload.num_hits; j++) {
    //         std::cout << "(" << random_payload.end_points[j].x << ", " << random_payload.end_points[j].y << ", " << random_payload.end_points[j].z << ") ";
    //     }

    //     std::cout << std::endl;
    //     // print ground truth pixel color
    //     std::cout << "Ground Truth Pixel Color: (" << random_payload.pixel_color_gt.x << ", " << random_payload.pixel_color_gt.y << ", " << random_payload.pixel_color_gt.z << ")" << std::endl;
    //     std::cout << std::endl << std::endl;
    // }
    

    int* h_batch_num_hits = (int*)malloc(batch_size * sizeof(int));
    float3* h_gt_pixels = (float3*)malloc(batch_size * sizeof(float3));
    h_view_dir = (float2*)malloc(batch_size * sizeof(float2));

    float* d_gt_pixels;
    
    int* d_batch_num_hits;
    CUDA_CHECK(hipMalloc((void **)&d_gt_pixels, batch_size * sizeof(float3)));
    CUDA_CHECK(hipMalloc((void **)&d_batch_num_hits, batch_size * sizeof(int)));
    CUDA_CHECK(hipMalloc((void **)&d_view_dir, batch_size * sizeof(float2)));

    int* d_batch_hit_inds;
    // We train our neural network for a specific amount of epochs
    for (int j = 0; j < num_epochs; ++j) {
        std::printf("Started training loop epoch %d\n", j);
        // shuffle ray payloads
        std::random_shuffle(ray_payloads.begin(), ray_payloads.end());
        // Loop through each set of images and poses in our training dataset
        for(int i = 0; i < ray_payloads.size(); i+=batch_size) {
            // get batch_size ray payloads from ray_payloads
            // std::cout << "Getting batch of ray payloads \n";
            std::vector<RayPayload> batch_ray_payloads(ray_payloads.begin() + i, ray_payloads.begin() + i + batch_size);
            // store num_hits in ray payloads in h_batch_num_hits

            // std::cout << "Going from AOS to SOA \n";
            for(int k = 0; k < batch_size; k++) {
                h_batch_num_hits[k] = batch_ray_payloads[k].num_hits;
                h_gt_pixels[k] = batch_ray_payloads[k].pixel_color_gt;
                h_view_dir[k] = batch_ray_payloads[k].view_dir;
            }
            CUDA_CHECK(hipMemcpyAsync(d_batch_num_hits, h_batch_num_hits, batch_size * sizeof(int), hipMemcpyHostToDevice));
            CUDA_CHECK(hipMemcpyAsync(d_gt_pixels, h_gt_pixels, batch_size * sizeof(float3), hipMemcpyHostToDevice));
            CUDA_CHECK(hipMemcpyAsync(d_view_dir, h_view_dir, batch_size * sizeof(float2), hipMemcpyHostToDevice));
            // turn d_batch_num_hits into a thrust device pointer
            thrust::device_ptr<int> dev_ptr_num_hits(d_batch_num_hits);
            int num_points = thrust::reduce(dev_ptr_num_hits, dev_ptr_num_hits + batch_size);
            std::cout << "num_points: " << num_points << std::endl;
            thrust::device_vector<int> d_hit_indsV(batch_size);
            thrust::exclusive_scan(dev_ptr_num_hits, dev_ptr_num_hits + batch_size, d_hit_indsV.begin());
            d_batch_num_hits = dev_ptr_num_hits.get();
            d_batch_hit_inds = thrust::raw_pointer_cast(d_hit_indsV.data());
            // print d_batch_num_hits and d_batch_hit_inds
            // std::cout << "Printing d_batch_num_hits and d_batch_hit_inds \n";
            // print_int_arr<<<1,1>>>(d_batch_num_hits, batch_size);
            // CUDA_CHECK(hipDeviceSynchronize());
            // print_int_arr<<<1,1>>>(d_batch_hit_inds, batch_size);
            // CUDA_CHECK(hipDeviceSynchronize());

            //free both
            float3* h_start_points = (float3*)malloc(num_points * sizeof(float3));
            float3* h_end_points = (float3*)malloc(num_points * sizeof(float3));
            // float* h_t_end = (float*)malloc(num_points * sizeof(float));

            std::cout << "Filling in start_points, end_points, and t_end \n";
            // fill in start_points, end_points, and t_end
            int offset = 0;
            for(int k = 0; k < batch_size; k++) {
                for(int l = 0; l < batch_ray_payloads[k].num_hits; l++) {
                    h_start_points[offset + l] = batch_ray_payloads[k].start_points[l];
                    h_end_points[offset + l] = batch_ray_payloads[k].end_points[l];
                    // h_t_end[offset + l] = batch_ray_payloads[k].t_end[l];
                }
                offset += batch_ray_payloads[k].num_hits;
            }

            std::cout << "Allocating GPU Buffers for Sampling \n";
            float3* d_start_points;
            float3* d_end_points;
            // float* d_t_end;
            //cudafree both
            CUDA_CHECK(hipMalloc((void **)&d_start_points, num_points * sizeof(float3)));
            CUDA_CHECK(hipMalloc((void **)&d_end_points, num_points * sizeof(float3)));
            // CUDA_CHECK(hipMalloc((void **)&d_t_end, num_points * sizeof(float)));

            std::cout << "Copying start_points, end_points, and t_end to GPU \n";
            CUDA_CHECK(hipMemcpyAsync(d_start_points, h_start_points, num_points * sizeof(float3), hipMemcpyHostToDevice));
            CUDA_CHECK(hipMemcpyAsync(d_end_points, h_end_points, num_points * sizeof(float3), hipMemcpyHostToDevice));
            // CUDA_CHECK(hipMemcpyAsync(d_t_end, h_t_end, num_points * sizeof(float), hipMemcpyHostToDevice));
            
            CUDA_CHECK(hipDeviceSynchronize());
            int samples_per_intersect = 32;
            printf("num_hits_cu: %d\n", num_points);
            int num_sampled_points = samples_per_intersect * num_points;
            printf("sampled_points: %d\n", num_sampled_points);
            num_sampled_points = (num_sampled_points / 256) * 256 + 256;
            printf("upsampled_points: %d\n", num_sampled_points);
            float* d_sampled_points;
            float* d_sampled_points_radiance;
            float* d_t_vals;
            unsigned int size_input = num_sampled_points * sizeof(float) * 5;
            unsigned int size_output = num_sampled_points * sizeof(float) * 4;
            printf("ALLOCATING %d bytes for samples (shouldn't be zero) \n", size_input);
            printf("ALLOCATING %d bytes for radiance (shouldn't be zero) \n", size_output);
            // cudafree all of these
            CUDA_CHECK(hipMalloc((void**)&d_sampled_points, size_input));
            CUDA_CHECK(hipMalloc((void**)&d_sampled_points_radiance,
                        size_output));
            CUDA_CHECK(hipMalloc((void**)&d_t_vals, sizeof(float) * num_sampled_points));
            

            // std::cout << "Printing start_points and end_points \n";
            // print_float3_arr<<<1,1>>>(d_start_points, num_points);
            // CUDA_CHECK(hipDeviceSynchronize());
            // print_float3_arr<<<1,1>>>(d_end_points, num_points);
            // CUDA_CHECK(hipDeviceSynchronize());

            // std::cout << "Launching Sampling Kernel \n";
            launchSampler(
                d_start_points,
                d_end_points,
                d_view_dir,
                d_t_vals,
                d_sampled_points,
                batch_size, grid_resolution,
                d_batch_num_hits, d_batch_hit_inds,
                SAMPLING_REGULAR, inference_stream);
            
            
            uint32_t padded_output_width = model.network->padded_output_width();
            tcnn::GPUMatrix<float> input_batch(n_input_dims, num_sampled_points);
            tcnn::GPUMatrix<tcnn::network_precision_t> output_fwd(padded_output_width, num_sampled_points);
            
            // printGPUMem();
            // printf("Launching Forward Pass\n");
            auto ctx = model.network->forward(inference_stream, input_batch, &output_fwd, true, true);
            // printf("Done Forward Pass\n");
            tcnn::GPUMatrix<tcnn::network_precision_t> output_slice = output_fwd.slice_rows(0, n_output_dims);
            
            int num_el = output_slice.n_elements();
            int blockSize1 = 1024;
            int numBlocks1 = (num_el + blockSize1 - 1) / blockSize1;
            convertHalfToFloat<<<numBlocks1,blockSize1>>>(output_slice.data(), d_sampled_points_radiance, num_el);
            // print radiance buffer values
            // printf("Printing radiance buffer values\n");
            // print_float4_arr<<<1,1>>>(d_sampled_points_radiance, num_sampled_points);
            // CUDA_CHECK(hipDeviceSynchronize());
            
            // Launch Volume Rendering kernel
            // printf("Launching Volume Rendering Kernel\n");
            
            // TODO: inference stream
            launch_volrender_cuda(
                d_sampled_points,
                d_sampled_points_radiance,
                d_batch_num_hits,
                d_batch_hit_inds,
                d_t_vals,
                batch_size,
                samples_per_intersect,
                d_pixels
            );
            // printf("Done Volume Rendering Kernel\n");
            // print pixel buffer values
            // printf("Printing pixel buffer values\n");
            // print_float_arr<<<1,1>>>(d_pixels, batch_size);
            // CUDA_CHECK(hipDeviceSynchronize());
            int blockSize2 = 1024;
            int numBlocks2 = (batch_size + blockSize2 - 1) / blockSize2;
            floatToHalf<<<numBlocks2, blockSize2>>>(d_pixels, d_pixels_half, batch_size);
            tcnn::GPUMatrix<tcnn::network_precision_t> predicted_image(d_pixels_half, batch_size, channels);
            tcnn::GPUMatrix<float> target_image(d_gt_pixels, batch_size, channels);
            tcnn::GPUMatrix<float> values(batch_size, channels);
            tcnn::GPUMatrix<tcnn::network_precision_t> gradients(batch_size, channels);
            model.loss->evaluate(1.0f, predicted_image, target_image, values, gradients);
            float batch_loss = tcnn::reduce_sum(values.data(), values.n_elements(), inference_stream);
            std::cout << "Batch Loss: " << batch_loss << std::endl;
            
            
            tcnn::network_precision_t* d_loss_mlp;
            CUDA_CHECK(hipMalloc((void**)&d_loss_mlp, sizeof(tcnn::network_precision_t) * 16 * num_sampled_points));
            
            launch_volrender_backward_cuda(
                values.data(),
                gradients.data(),
                d_sampled_points_radiance,
                d_t_vals,
                d_batch_num_hits,
                d_batch_hit_inds,
                batch_size,
                samples_per_intersect,
                d_loss_mlp
            );
            // printf("Done Volume Rendering Backward Kernel\n");
            tcnn::GPUMatrix<tcnn::network_precision_t> loss_mlp(d_loss_mlp, 16, num_sampled_points);
            model.network->backward(inference_stream, *ctx, input_batch, output_fwd, loss_mlp);
            printGPUMem();
            // free buffers
            hipFree(d_sampled_points);
            hipFree(d_sampled_points_radiance);
            hipFree(d_t_vals);
            hipFree(d_start_points);
            hipFree(d_end_points);
            hipFree(d_loss_mlp);
            free(h_start_points);
            free(h_end_points);
            // std::cout << "Done freeing buffers \n";
            printGPUMem();
        }
        break;
    }
    return 0;
}
