#include "hip/hip_runtime.h"
#include <iostream>
#include <string>
#include <vector>
#include <cstdio>
#include <fstream>
#include <math.h>
#include "stdio.h"
#include "sampler.h"

#include "optix_function_table_definition.h"
#include "optix_stubs.h"
#include "optix.h"
#include "optix_types.h"

#define STB_IMAGE_IMPLEMENTATION
#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "stb_image.h"
#include "stb_image_write.h"
#include "tiny-cuda-nn/common.h"
#include "tiny-cuda-nn/gpu_matrix.h"
#include <json/json.h>
#include "rtx/include/params.h"
#include "rtx/include/rtxFunctions.h"

#include "data_loader.h"

// Configure the model
nlohmann::json config = {
	{"loss", {
		{"otype", "L2"}
	}},
	{"optimizer", {
		{"otype", "Adam"},
		{"learning_rate", 1e-3},
	}},
	{"encoding", {
		{"otype", "HashGrid"},
		{"n_levels", 16},
		{"n_features_per_level", 2},
		{"log2_hashmap_size", 19},
		{"base_resolution", 16},
		{"per_level_scale", 2.0},
	}},
	{"network", {
		{"otype", "FullyFusedMLP"},
		{"activation", "ReLU"},
		{"output_activation", "None"},
		{"n_neurons", 64},
		{"n_hidden_layers", 2},
	}},
};



__global__ void printFloats(float* gpuPointer, int size)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    if (tid < size)
    {
        printf("Value at index %d: %f\n", tid, gpuPointer[tid]);
    }
}
__global__ void print_batch(float* batch, int batch_size, int image_size) {
    //printf("HELLO???\n");
    // int tid = blockIdx.x * blockDim.x + threadIdx.x;
    // if (tid < batch_size) {
    //     printf("Batch %d\n", tid);
    //     for (int i = 0; i < image_size; ++i) {
    //         printf("%f ", batch[tid * image_size + i]);
    //     }
    //     printf("\n");
    // }
}

__global__ void gatherIntersections(
    float3* d_start_points, 
    float3* d_end_points, 
    int* d_num_hits, 
    float3* d_intersect_start,
    float3* d_intersect_end,
    int width, int height, int grid_size)
{
    // Calculate the index of the pixel this thread should process.
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < width && y < height)
    {
        // Calculate the base index for this pixel in the d_start_points and d_end_points arrays.
        int base_index = (y * width + x) * grid_size;

        // Find the number of grid cells hit by the ray from this pixel.
        int num_hits = d_num_hits[y * width + x];

        // For each hit, gather the entry and exit points.
        for (int i = 0; i < num_hits; ++i)
        {
            float3 start_point = d_start_points[base_index + i];
            float3 end_point = d_end_points[base_index + i];

            // Store the intersection points.
            d_intersect_start[2 * (base_index + i)] = start_point;
            d_intersect_end[2 * (base_index + i)] = end_point;
        }
    }
}
 
// Creates a grid of Axis-aligned bounding boxes with specified resolution
// Bounding box coordinates are specified in normalized coordinates from -1 to 1
// TODO: make this a CUDA kernel
std::vector<OptixAabb> make_grid(int resolution) {
    std::vector<OptixAabb> grid;
    float box_length = 2.0f/ (float)resolution;
    for(int x = 0; x < resolution; x++) {
        for(int y = 0; y < resolution; y++) {
            for(int z = 0; z < resolution; z++) {
                OptixAabb aabb;
                aabb.minX = -1.0f + (float)x * box_length;
                aabb.maxX = -1.0f + x * box_length + box_length;
                aabb.minY = -1.0f + y * box_length;
                aabb.maxY = -1.0f + y * box_length + box_length;
                aabb.minZ = -1.0f + z * box_length;
                aabb.maxZ = -1.0f + z * box_length + box_length;
                grid.push_back(aabb);
                //std::printf("aabb (%.2f %.2f %.2f) (%.2f %.2f %.2f)\n",
                //        aabb.minX, aabb.minY, aabb.minZ, aabb.maxX, aabb.maxY, aabb.maxZ);
            }
        }
    }
    return grid;
}
//auto model = tcnn::create_from_config(n_input_dims, n_output_dims, config);

#define EPOCHS 10
#define BATCH_SIZE tcnn::batch_size_granularity
#define DATASET_SIZE 1000

RTXDataHolder *rtx_dataholder;

__global__ void print_intersections(float3* start, float3* end, int* num_hits, int num_prim) {
    printf("Intersections\n");
    for (int i = 0; i < 10; ++i) {
        printf("ray (%i): %i hits\n", i, num_hits[i]); // origin = (%.2f, %.2f, %.2f)\n  ",
        for (int j = 0; j < num_hits[i]; ++j) {
            float3 s = start[i*num_prim + j];
            float3 e = end[i*num_prim + j];
            printf("   (%.2f %.2f %.2f) (%.2f %.2f %.2f)\n", s.x, s.y, s.z, e.x, e.y, e.z);
        }
    }
}



int main() {
    // load data from files
    // TODO: take images and poses from json and load into DataLoader
    int num_epochs = EPOCHS;
    std::cout << "---------------------- Loading Data ----------------------\n";
    // Loads the Training, validation, and test sets from the synthetic lego scene
    std::vector<ImageDataset> datasets = load_data(SceneType::SYNTHETIC, SyntheticName::LEGO);
    auto train_set = datasets[0];
    unsigned int width = train_set.image_width;
    unsigned int height = train_set.image_height;
    unsigned int channels = train_set.image_channels;
    float training_focal = train_set.focal;
    float aspect_ratio = (float)width / (float)height;
    float focal_length = 1.0f / tan(0.5f * training_focal);
    size_t image_size = width * height * channels;
    // get training dataset from datasets
    std::vector<float*> training_images = datasets[0].images;
    std::vector<float*> training_poses = datasets[0].poses;
    std::cout << "---------------------- Data Loaded ----------------------\n\n\n";
    // Initialize our Optix Program Groups and Pipeline
    // We also build our initial dense acceleration structure of AABBs

    std::cout << "---------------------- Initializing Optix ----------------------\n";
    hipStream_t inference;
    hipStream_t training;
    CUDA_CHECK(hipStreamCreate(&inference));
    CUDA_CHECK(hipStreamCreate(&training));
    std::string ptx_filename = BUILD_DIR "bin/ptx/optixPrograms.ptx";

    rtx_dataholder = new RTXDataHolder();
    std::cout << "Initializing Context \n";
    rtx_dataholder->initContext();
    std::cout << "Reading PTX file and creating modules \n";
    rtx_dataholder->createModule(ptx_filename);
    std::cout << "Creating Optix Program Groups \n";
    rtx_dataholder->createProgramGroups();
    std::cout << "Linking Pipeline \n";
    rtx_dataholder->linkPipeline(false);
    std::cout << "Building Shader Binding Table (SBT) \n";
    rtx_dataholder->buildSBT();
    
    // Build our initial dense acceleration structure
    int grid_resolution = 8;
    std::cout << "Building Acceleration Structure \n";
    std::vector<OptixAabb> grid = make_grid(grid_resolution);
    int num_primitives = grid.size();
    
    OptixAabb* d_aabb = rtx_dataholder->initAccelerationStructure(grid);
    std::cout << "Done Building Acceleration Structure \n";
    std::cout << "---------------------- Done Initializing Optix ----------------------\n\n\n";

    std::cout << "Allocating Buffers on GPU" << std::endl;
    float* d_image, *d_look_at;
    CUDA_CHECK(hipMalloc((void **)&d_image, image_size * sizeof(float)));
    CUDA_CHECK(hipMalloc((void **)&d_look_at, 16 * sizeof(float)));
    std::cout << "Image Buffers Allocated on GPU" << std::endl;
    // Allocate buffers to hold outputs from ray intersection tests
    // start and end points are equal to # of AABBs in AS per ray [width * height * num_primitives]
    float3 *d_start_points;
    float3 *d_end_points;
    int *d_num_hits;
            
    CUDA_CHECK(hipMalloc((void **)&d_start_points, width * height * 3 * grid_resolution * sizeof(float3)));
    CUDA_CHECK(hipMalloc((void **)&d_end_points, width * height * 3 * grid_resolution * sizeof(float3)));
    CUDA_CHECK(hipMalloc((void **)&d_num_hits, width * height * sizeof(int)));
    std::cout << "Ray Intersection Buffers Allocated on GPU" << std::endl;

    Params *d_param;
    CUDA_CHECK(hipMalloc((void **)&d_param, sizeof(Params)));
    std::cout << "Params Buffer Allocated on GPU" << std::endl;


    // We train our neural network for a specific amount of epochs
    for (int j = 0; j < num_epochs; ++j) {
        std::printf("Started training loop epoch %d\n", j);
        
        // Loop through each set of images and poses in our training dataset
        for(int i = 0; i < training_images.size(); i++) {
            float* image = training_images[i];
            float* look_at = training_poses[i];

            // transfer image and look_at to GPU
            CUDA_CHECK(hipMemcpyAsync(d_image, image, image_size * sizeof(float), hipMemcpyHostToDevice, inference));
            CUDA_CHECK(hipMemcpyAsync(d_look_at, look_at, 16 * sizeof(float), hipMemcpyHostToDevice, inference));

            // Memset ray intersection buffers
            CUDA_CHECK(hipMemset(d_start_points, -2, width * height * 3 * grid_resolution * sizeof(float3)));
            CUDA_CHECK(hipMemset(d_end_points, -2, width * height * 3 * grid_resolution * sizeof(float3)));
            CUDA_CHECK(hipMemset(d_num_hits, 0, width * height * sizeof(int)));

            // Algorithmic parameters and data pointers used in GPU program
            Params params;
            // params.transform_matrix = transform_matrix;
            float d =  2.0f / grid_resolution;
            params.delta = make_float3(d, d, d);
            params.min_point = make_float3(-1, -1, -1);
            params.max_point = make_float3(1, 1, 1);
            params.intersection_arr_size = 3 * grid_resolution;
            params.width = width;
            params.height = height;
            params.focal_length = focal_length;
            params.aspect_ratio = aspect_ratio;
            params.handle = rtx_dataholder->gas_handle;
            params.aabb = d_aabb;
            params.start_points = d_start_points;
            params.end_points = d_end_points;
            params.num_hits = d_num_hits;
            params.num_primitives = num_primitives;
            params.look_at = d_look_at;

            
            CUDA_CHECK(hipMemcpy(d_param, &params, sizeof(params), hipMemcpyHostToDevice));
            const OptixShaderBindingTable &sbt_ray_march = rtx_dataholder->sbt_ray_march;
            std::cout << "Launching Ray Tracer in Ray Marching Mode (" << width*height << " rays)\n";
            OPTIX_CHECK(optixLaunch(rtx_dataholder->pipeline_ray_march, inference,
                                    reinterpret_cast<hipDeviceptr_t>(d_param),
                                    sizeof(Params), &sbt_ray_march, width, height, 1));
            CUDA_CHECK(hipStreamSynchronize(inference));

            // CUDA Launch Sampling Kernel given entry and exit points from this perspective
            d_start_points = params.start_points;
            d_end_points = params.end_points;
            d_num_hits = params.num_hits;

            print_intersections<<<1,1>>>(d_start_points, d_end_points, d_num_hits, 3 * grid_resolution);
            CUDA_CHECK(hipDeviceSynchronize());

            std::cout << "Launching Sampling Kernel \n";
            //each point stores a location xyz and a viewing direction phi and psi
            float5* d_sampled_points;
            int num_points;
            int samples_per_intersect = 32;
            std::cout << "Finding number of points to sample \n";
            int* h_num_hits = (int*)malloc(sizeof(int) * width * height);

            hipMemcpy(h_num_hits, d_num_hits, sizeof(int) * width * height, hipMemcpyDeviceToHost);
            int num_hits = 0;
            for (int i = 0; i < width * height; ++i) {
                num_hits += h_num_hits[i];
            }
            printf("num_hits: %d\n", num_hits);
            printf("sampled_points: %d\n", samples_per_intersect * num_hits);
            int size_samples = num_hits * samples_per_intersect * sizeof(float5);
            printf("ALLOCATING %d bytes for samples (shouldn't be zero) \n", size_samples);

            CUDA_CHECK(hipMalloc((void**)&d_sampled_points, size_samples));

            launchUniformSampler(
                d_start_points,
                d_end_points,
                d_num_hits,
                d_sampled_points,
                samples_per_intersect,
                width, height,
                num_primitives, 
                inference,
                num_points);

            // tcnn inference on point buffer from sampling kernels
            
            // Optix Launch Volume Rendering kernel

            // tcnn compute loss and backpropagate

	    break;
        }
        break;
    }
    return 0;
}
