#include "hip/hip_runtime.h"
#include <iostream>
#include <string>
#include <vector>
#include <cstdio>
#include <fstream>

#include "stdio.h"


#include "optix_function_table_definition.h"
#include "optix_stubs.h"
#include "optix.h"
#include "optix_types.h"

#define STB_IMAGE_IMPLEMENTATION
#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "stb_image.h"
#include "stb_image_write.h"
#include "tiny-cuda-nn/common.h"
#include "tiny-cuda-nn/gpu_matrix.h"
#include <json/json.h>
#include "rtx/include/params.h"
#include "rtx/include/rtxFunctions.h"

#include "data_loader.h"
// #include "transform_loader.h"

// Configure the model
nlohmann::json config = {
	{"loss", {
		{"otype", "L2"}
	}},
	{"optimizer", {
		{"otype", "Adam"},
		{"learning_rate", 1e-3},
	}},
	{"encoding", {
		{"otype", "HashGrid"},
		{"n_levels", 16},
		{"n_features_per_level", 2},
		{"log2_hashmap_size", 19},
		{"base_resolution", 16},
		{"per_level_scale", 2.0},
	}},
	{"network", {
		{"otype", "FullyFusedMLP"},
		{"activation", "ReLU"},
		{"output_activation", "None"},
		{"n_neurons", 64},
		{"n_hidden_layers", 2},
	}},
};
__global__ void printFloats(float* gpuPointer, int size)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    if (tid < size)
    {
        printf("Value at index %d: %f\n", tid, gpuPointer[tid]);
    }
}
__global__ void print_batch(float* batch, int batch_size, int image_size) {
    //printf("HELLO???\n");
    // int tid = blockIdx.x * blockDim.x + threadIdx.x;
    // if (tid < batch_size) {
    //     printf("Batch %d\n", tid);
    //     for (int i = 0; i < image_size; ++i) {
    //         printf("%f ", batch[tid * image_size + i]);
    //     }
    //     printf("\n");
    // }
}
// Creates a grid of Axis-aligned bounding boxes with specified resolution
// Bounding box coordinates are specified in normalized coordinates from -1 to 1
std::vector<OptixAabb> make_grid(int resolution) {
    std::vector<OptixAabb> grid;
    float box_length = 2.0f/ (float)resolution;
    for(int x = 0; x < resolution; x++) {
        for(int y = 0; y < resolution; y++) {
            for(int z = 0; z < resolution; z++) {
                OptixAabb aabb;
                aabb.minX = -1.0f + (float)x * box_length;
                aabb.maxX = -1.0f + x * box_length + box_length;
                aabb.minY = -1.0f + y * box_length;
                aabb.maxY = -1.0f + y * box_length + box_length;
                aabb.minZ = -1.0f + z * box_length;
                aabb.maxZ = -1.0f + z * box_length + box_length;
                grid.push_back(aabb);
            }
        }
    }
    return grid;
}
//auto model = tcnn::create_from_config(n_input_dims, n_output_dims, config);

#define EPOCHS 10
#define BATCH_SIZE tcnn::batch_size_granularity
#define DATASET_SIZE 1000

RTXDataHolder *rtx_dataholder;




int main() {
    // load data from files
    // TODO: take images and poses from json and load into DataLoader
    int num_epochs = EPOCHS;
    // Loads the Training, validation, and test sets from the synthetic lego scene
    std::vector<ImageDataset> datasets = load_data(SceneType::SYNTHETIC, SceneName::LEGO);
    unsigned int width = datasets[0].image_width;
    unsigned int height = datasets[0].image_height;
    unsigned int channels = datasets[0].image_channels;
    size_t image_size = width * height * channels;
    // get training dataset from datasets
    std::vector<float*> training_images = datasets[0].images;
    std::vector<float*> training_poses = datasets[0].poses;
    
    // Initialize our Optix Program Groups and Pipeline
    // We also build our initial dense acceleration structure of AABBs
    hipStream_t stream;
    CUDA_CHECK(hipStreamCreate(&stream));
    std::string ptx_filename = BUILD_DIR "/ptx/optixPrograms.ptx";

    rtx_dataholder = new RTXDataHolder();
    std::cout << "Initializing Context \n";
    rtx_dataholder->initContext();
    std::cout << "Reading PTX file and creating modules \n";
    rtx_dataholder->createModule(ptx_filename);
    std::cout << "Creating Optix Program Groups \n";
    rtx_dataholder->createProgramGroups();
    std::cout << "Linking Pipeline \n";
    rtx_dataholder->linkPipeline(false);
    std::cout << "Building Shader Binding Table (SBT) \n";
    rtx_dataholder->buildSBT();
    
    // Build our initial dense acceleration structure
    int grid_resolution = 8;
    std::cout << "Building Acceleration Structure \n";
    std::vector<OptixAabb> grid = make_grid(grid_resolution);
    int num_primitives = grid.size();
    
    rtx_dataholder->initAccelerationStructure(grid);
    std::cout << "Done Building Acceleration Structure \n";
    // We train our neural network for a specific amount of epochs
    for (int j = 0; j < num_epochs; ++j) {
        std::printf("Started training loop epoch %d\n", j);
        
        // Loop through each set of images and poses in our training dataset
        for(int i = 0; i < training_images.size(); i++) {
            float* image = training_images[i];
            float* look_at = training_poses[i];

            float* d_image, *d_look_at;
            // allocate memory for image and look_at on GPU
            CUDA_CHECK(hipMalloc((void **)&d_image, image_size * sizeof(float)));
            CUDA_CHECK(hipMalloc((void **)&d_look_at, 16 * sizeof(float)));

            // transfer image and look_at to GPU
            CUDA_CHECK(hipMemcpyAsync(d_image, image, image_size * sizeof(float), hipMemcpyHostToDevice, stream));
            CUDA_CHECK(hipMemcpyAsync(d_look_at, look_at, 16 * sizeof(float), hipMemcpyHostToDevice, stream));

            std::cout << "Allocating Buffers for Ray Intersection Tests \n";
            // Allocate buffers to hold outputs from ray intersection tests
            // start and end points are equal to # of AABBs in AS per ray [width * height * num_primitives]
            float3 *d_start_points;
            float3 *d_end_points;
            int *d_num_hits;
            CUDA_CHECK(hipMalloc(
                (void **)&d_start_points, width * height * num_primitives * sizeof(float3)));
            CUDA_CHECK(hipMalloc(
                (void **)&d_end_points, width * height * num_primitives * sizeof(float3)));
            CUDA_CHECK(hipMalloc((void **)&d_num_hits, width * height * sizeof(int)));
            CUDA_CHECK(hipMemset(d_start_points, -2, width * height * num_primitives * sizeof(float3)));
            CUDA_CHECK(hipMemset(d_end_points, -2, width * height * num_primitives * sizeof(float3)));
            CUDA_CHECK(hipMemset(d_num_hits, 0, width * height * sizeof(int)));

            // Algorithmic parameters and data pointers used in GPU program
            Params params;
            // params.transform_matrix = transform_matrix;
            float d =  2.0f / grid_resolution;
            params.delta = make_float3(d, d, d);
            params.min_point = make_float3(-1, -1, -1);
            params.max_point = make_float3(1, 1, 1);
            params.width = width;
            params.height = height;
            params.handle = rtx_dataholder->gas_handle;
            params.start_points = d_start_points;
            params.end_points = d_end_points;
            params.num_hits = d_num_hits;
            params.num_primitives = num_primitives;
            Params *d_param;
            CUDA_CHECK(hipMalloc((void **)&d_param, sizeof(Params)));
            CUDA_CHECK(hipMemcpy(d_param, &params, sizeof(params), hipMemcpyHostToDevice));
            const OptixShaderBindingTable &sbt_ray_march = rtx_dataholder->sbt_ray_march;
            std::cout << "Launching Ray Tracer in Ray Marching Mode \n";
            OPTIX_CHECK(optixLaunch(rtx_dataholder->pipeline_ray_march, stream,
                                    reinterpret_cast<hipDeviceptr_t>(d_param),
                                    sizeof(Params), &sbt_ray_march, width, height, 1));
            CUDA_CHECK(hipStreamSynchronize(stream));

            // CUDA Launch Sampling Kernel given entry and exit points from this perspective
            d_start_points = params.start_points;
            d_end_points = params.end_points;
            d_num_hits = params.num_hits;
            std::cout << "Launching Sampling Kernel \n";
            int num_points = 0;
            // tcnn inference on point buffer from sampling kernels
            
            // Optix Launch Volume Rendering kernel

            // tcnn compute loss and backpropagate


        }
    }
    return 0;
}
