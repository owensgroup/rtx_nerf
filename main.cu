#include "hip/hip_runtime.h"
#include <iostream>
#include <cmath>
#include <string>
#include <vector>
#include <cstdio>
#include <fstream>
#include <math.h>
#include <thrust/device_vector.h>
#include <thrust/device_ptr.h>
#include "stdio.h"
#include "sampler.h"

#include "optix_function_table_definition.h"
#include "optix_stubs.h"
#include "optix.h"
#include "optix_types.h"

#define STB_IMAGE_IMPLEMENTATION
#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "stb_image.h"
#include "stb_image_write.h"
#include "tiny-cuda-nn/common.h"
#include "tiny-cuda-nn/gpu_matrix.h"
#include "tiny-cuda-nn/config.h"
#include "tiny-cuda-nn/reduce_sum.h"
#include <json/json.h>
#include "rtx/include/params.h"
#include "rtx/include/rtxFunctions.h"

#include "data_loader.h"
#include "vol_render.h"
// Configure the model
nlohmann::json config = {
	{"loss", {
		{"otype", "L2"}
	}},
    // adam optimizer decays from 5e-4 to 5e-5
	{"optimizer", {
		{"otype", "Adam"},
		{"learning_rate", 1e-3},
        {"beta1", 0.9},
        {"beta2", 0.999},
        {"epsilon", 1e-8}
	}},
	{"encoding", {
        {"otype", "Composite"},
        {"nested", {
            {
                {"n_dims_to_encode", 3}, // Spatial dims
                {"otype", "Frequency"},
                {"n_frequencies", 10}
            },
            {
                {"n_dims_to_encode", 2}, // Non-linear appearance dims.
                {"otype", "Frequency"},
                {"n_bins", 4}
            }
        }}
    }},
	{"network", {
		{"otype", "FullyFusedMLP"},
		{"activation", "ReLU"},
		{"output_activation", "Sigmoid"},
		{"n_neurons", 128},
		{"n_hidden_layers", 8}
	}}
};

template<typename T>
void printGPUMatrix(
    const tcnn::GPUMatrix<T>& matrix,
    int n_rows, int n_cols) {
    // Get the dimensions of the matrix
    uint32_t rows = matrix.rows();
    uint32_t cols = matrix.cols();

    // Allocate host memory to store the matrix data
    T* hostData = new T[rows * cols];

    // Copy the matrix data from GPU to host
    hipMemcpy(hostData, matrix.data(), sizeof(T) * rows * cols, hipMemcpyDeviceToHost);

    // Print the matrix values
    for (uint32_t i = 0; i < n_rows; i++) {
        for (uint32_t j = 0; j < n_cols; j++) {
            std::cout << hostData[i * cols + j] << " ";
        }
        std::cout << std::endl;
    }

    // Free the host memory
    delete[] hostData;
}

__global__ void printFloats(float* gpuPointer, int size)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    if (tid < size)
    {
        printf("Value at index %d: %f\n", tid, gpuPointer[tid]);
    }
}
__global__ void print_batch(float* batch, int batch_size, int image_size) {
    //printf("HELLO???\n");
    // int tid = blockIdx.x * blockDim.x + threadIdx.x;
    // if (tid < batch_size) {
    //     printf("Batch %d\n", tid);
    //     for (int i = 0; i < image_size; ++i) {
    //         printf("%f ", batch[tid * image_size + i]);
    //     }
    //     printf("\n");
    // }
}

__global__ void gatherIntersections(
    float3* d_start_points, 
    float3* d_end_points, 
    int* d_num_hits, 
    float3* d_intersect_start,
    float3* d_intersect_end,
    int width, int height, int grid_size)
{
    // Calculate the index of the pixel this thread should process.
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < width && y < height)
    {
        // Calculate the base index for this pixel in the d_start_points and d_end_points arrays.
        int base_index = (y * width + x) * grid_size;

        // Find the number of grid cells hit by the ray from this pixel.
        int num_hits = d_num_hits[y * width + x];

        // For each hit, gather the entry and exit points.
        for (int i = 0; i < num_hits; ++i)
        {
            float3 start_point = d_start_points[base_index + i];
            float3 end_point = d_end_points[base_index + i];

            // Store the intersection points.
            d_intersect_start[2 * (base_index + i)] = start_point;
            d_intersect_end[2 * (base_index + i)] = end_point;
        }
    }
}
 
// Creates a grid of Axis-aligned bounding boxes with specified resolution
// Bounding box coordinates are specified in normalized coordinates from -1 to 1
// TODO: make this a CUDA kernel
std::vector<OptixAabb> make_grid(int resolution) {
    std::vector<OptixAabb> grid;
    float box_length = 2.0f/ (float)resolution;
    for(int x = 0; x < resolution; x++) {
        for(int y = 0; y < resolution; y++) {
            for(int z = 0; z < resolution; z++) {
                OptixAabb aabb;
                aabb.minX = -1.0f + (float)x * box_length;
                aabb.maxX = -1.0f + x * box_length + box_length;
                aabb.minY = -1.0f + y * box_length;
                aabb.maxY = -1.0f + y * box_length + box_length;
                aabb.minZ = -1.0f + z * box_length;
                aabb.maxZ = -1.0f + z * box_length + box_length;
                grid.push_back(aabb);
                //std::printf("aabb (%.2f %.2f %.2f) (%.2f %.2f %.2f)\n",
                //        aabb.minX, aabb.minY, aabb.minZ, aabb.maxX, aabb.maxY, aabb.maxZ);
            }
        }
    }
    return grid;
}
//auto model = tcnn::create_from_config(n_input_dims, n_output_dims, config);

#define EPOCHS 10
#define BATCH_SIZE tcnn::batch_size_granularity*2048
#define DATASET_SIZE 1000

RTXDataHolder *rtx_dataholder;

__global__ void print_intersections(float3* start, float3* end, int* num_hits, int num_prim) {
    printf("Intersections\n");
    for (int i = 0; i < 100; ++i) {
        printf("ray (%i): %i hits\n", i, num_hits[i]); // origin = (%.2f, %.2f, %.2f)\n  ",
        for (int j = 0; j < num_hits[i]; ++j) {
            float3 s = start[i*num_prim + j];
            float3 e = end[i*num_prim + j];
            printf("   (%.2f %.2f %.2f) (%.2f %.2f %.2f)\n", s.x, s.y, s.z, e.x, e.y, e.z);
        }
    }
}

__global__ void convertHalfToFloat(__half* input, float* output, int size) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < size) {
        output[tid] = __half2float(input[tid]);
    }
}

__global__ void floatToHalf(float* input, __half* output, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        output[idx] = __float2half(input[idx]);
    }
}

__global__ void print_int_arr(int* arr, int width, int height) {
    printf("Printing int array\n");
    for (int i = 0; i < 10; ++i) {
        for(int j = 0; j < 10; ++j) {
            printf("%d ", arr[i * width + j]);
        }
        printf("\n");
    }
    printf("\n");
}

__global__ void print_float_arr(float* arr, int size) {
    printf("Printing float array\n");
    for (int i = 0; i < 10; ++i) {
        printf("%f ", arr[i]);
    }
    printf("\n");
}

__global__ void print_float2_arr(float2* arr, int width, int height) {
    printf("Printing float2 array\n");
    for (int i = 0; i < 10; ++i) {
        for(int j = 0; j < 10; ++j) {
            printf("%f %f ", arr[i * width + j].x, arr[i * width + j].y);
        }
        printf("\n");
    }
    printf("\n");
}

__global__ void print_float5_arr(float* arr, int size) {
    printf("Printing first 32 points \n");
    for(int i = 0; i < 32; ++i) {
        printf("%f %f %f %f %f\n", arr[i*5], arr[i*5+1], arr[i*5+2], arr[i*5+3], arr[i*5+4]);
    }
    printf("\n");
    printf("Printing last 32 points \n");
    for(int i = size-32; i < size; ++i) {
        printf("%f %f %f %f %f\n", arr[i*5], arr[i*5+1], arr[i*5+2], arr[i*5+3], arr[i*5+4]);
    }
    printf("\n");
}

__global__ void print_float3_arr(float3* arr, int size) {
    printf("Printing float3 array\n");
    printf("Printing first 32 points \n");
    for(int i = 0; i < 32; ++i) {
        printf("%f %f %f\n", arr[i].x, arr[i].y, arr[i].z);
    }
    printf("\n");
    printf("Printing last 32 points \n");
    for(int i = size-32; i < size; ++i) {
        printf("%f %f %f\n", arr[i].x, arr[i].y, arr[i].z);
    }
    printf("\n");
}

__global__ void print_float4_arr(float* arr, int size) {
    printf("Printing float4 array\n");
    printf("Printing first 32 points \n");
    for(int i = 0; i < 32; ++i) {
        printf("%f %f %f %f\n", arr[i*4], arr[i*4+1], arr[i*4+2], arr[i*4+3]);
    }
    printf("\n");
    printf("Printing last 32 points \n");
    for(int i = size-32; i < size; ++i) {
        printf("%f %f %f %f\n", arr[i*4], arr[i*4+1], arr[i*4+2], arr[i*4+3]);
    }
    printf("\n");
}

int main() {
    // load data from files
    // TODO: take images and poses from json and load into DataLoader
    int n_input_dims = 5;
    int n_output_dims = 4;
    int batch_size = BATCH_SIZE;
    auto model = tcnn::create_from_config(n_input_dims, n_output_dims, config);
    int num_epochs = EPOCHS;
    std::cout << "---------------------- Loading Data ----------------------\n";
    // Loads the Training, validation, and test sets from the synthetic lego scene
    std::vector<ImageDataset> datasets = load_data(SceneType::SYNTHETIC, SyntheticName::LEGO);
    auto train_set = datasets[0];
    unsigned int width = train_set.image_width;
    unsigned int height = train_set.image_height;
    unsigned int channels = train_set.image_channels;
    printf("Channels: %i\n", channels);
    float training_focal = train_set.focal;
    float aspect_ratio = (float)width / (float)height;
    float focal_length = 1.0f / tan(0.5f * training_focal);
    size_t image_size = width * height * channels;
    // get training dataset from datasets
    std::vector<float*> training_images = datasets[0].images;
    std::vector<float*> training_poses = datasets[0].poses;
    std::cout << "---------------------- Data Loaded ----------------------\n\n\n";
    // Initialize our Optix Program Groups and Pipeline
    // We also build our initial dense acceleration structure of AABBs

    std::cout << "---------------------- Initializing Optix ----------------------\n";
    hipStream_t inference_stream;
    hipStream_t training_stream;
    CUDA_CHECK(hipStreamCreate(&inference_stream));
    CUDA_CHECK(hipStreamCreate(&training_stream));
    std::string ptx_filename = BUILD_DIR "bin/ptx/optixPrograms.ptx";

    rtx_dataholder = new RTXDataHolder();
    std::cout << "Initializing Context \n";
    rtx_dataholder->initContext();
    std::cout << "Reading PTX file and creating modules \n";
    rtx_dataholder->createModule(ptx_filename);
    std::cout << "Creating Optix Program Groups \n";
    rtx_dataholder->createProgramGroups();
    std::cout << "Linking Pipeline \n";
    rtx_dataholder->linkPipeline(false);
    std::cout << "Building Shader Binding Table (SBT) \n";
    rtx_dataholder->buildSBT();
    
    // Build our initial dense acceleration structure
    int grid_resolution = 8;
    std::cout << "Building Acceleration Structure \n";
    std::vector<OptixAabb> grid = make_grid(grid_resolution);
    int num_primitives = grid.size();
    
    OptixAabb* d_aabb = rtx_dataholder->initAccelerationStructure(grid);
    std::cout << "Done Building Acceleration Structure \n";
    std::cout << "---------------------- Done Initializing Optix ----------------------\n\n\n";

    std::cout << "Allocating Buffers on GPU" << std::endl;
    float* d_image, *d_look_at;
    CUDA_CHECK(hipMalloc((void **)&d_image, image_size * sizeof(float)));
    CUDA_CHECK(hipMalloc((void **)&d_look_at, 16 * sizeof(float)));
    std::cout << "Image Buffers Allocated on GPU" << std::endl;
    // Allocate buffers to hold outputs from ray intersection tests
    // start and end points are equal to # of AABBs in AS per ray [width * height * num_primitives]
    float3 *d_start_points;
    float3 *d_end_points;
    int *d_num_hits;
    float2 *d_view_dir;
    float* d_pixels;
    float* d_temp_out;
    tcnn::network_precision_t* d_pixels_half;
    CUDA_CHECK(hipMalloc((void**)&d_pixels, width * height * sizeof(float) * 3));
    CUDA_CHECK(hipMalloc((void **)&d_start_points, width * height * 3 * grid_resolution * sizeof(float3)));
    CUDA_CHECK(hipMalloc((void **)&d_end_points, width * height * 3 * grid_resolution * sizeof(float3)));
    CUDA_CHECK(hipMalloc((void **)&d_num_hits, width * height * sizeof(int)));
    CUDA_CHECK(hipMalloc((void **)&d_view_dir, width * height * sizeof(float2)));
    CUDA_CHECK(hipMalloc((void **)&d_temp_out, batch_size * n_output_dims * sizeof(float)));
    CUDA_CHECK(hipMalloc((void **)&d_pixels_half, width * height * sizeof(tcnn::network_precision_t) * 3));
    std::cout << "Ray Intersection Buffers Allocated on GPU" << std::endl;

    

    Params *d_param;
    CUDA_CHECK(hipMalloc((void **)&d_param, sizeof(Params)));
    std::cout << "Params Buffer Allocated on GPU" << std::endl;

    // We train our neural network for a specific amount of epochs
    for (int j = 0; j < num_epochs; ++j) {
        std::printf("Started training loop epoch %d\n", j);
        
        // Loop through each set of images and poses in our training dataset
        for(int i = 0; i < training_images.size(); i++) {
            float* image = training_images[i];
            float* look_at = training_poses[i];
            
            
            // transfer image and look_at to GPU
            CUDA_CHECK(hipMemcpyAsync(d_image, image, image_size * sizeof(float), hipMemcpyHostToDevice, inference_stream));
            CUDA_CHECK(hipMemcpyAsync(d_look_at, look_at, 16 * sizeof(float), hipMemcpyHostToDevice, inference_stream));

            // Memset ray intersection buffers
            CUDA_CHECK(hipMemset(d_start_points, -2, width * height * 3 * grid_resolution * sizeof(float3)));
            CUDA_CHECK(hipMemset(d_end_points, -2, width * height * 3 * grid_resolution * sizeof(float3)));
            CUDA_CHECK(hipMemset(d_num_hits, 0, width * height * sizeof(int)));

            // Algorithmic parameters and data pointers used in GPU program
            Params params;
            // params.transform_matrix = transform_matrix;
            float d =  2.0f / grid_resolution;
            params.delta = make_float3(d, d, d);
            params.min_point = make_float3(-1, -1, -1);
            params.max_point = make_float3(1, 1, 1);
            params.intersection_arr_size = 3 * grid_resolution;
            params.width = width;
            params.height = height;
            params.focal_length = focal_length;
            params.aspect_ratio = aspect_ratio;
            params.handle = rtx_dataholder->gas_handle;
            params.aabb = d_aabb;
            params.start_points = d_start_points;
            params.end_points = d_end_points;
            params.num_hits = d_num_hits;
            params.num_primitives = num_primitives;
            params.look_at = d_look_at;
            params.viewing_direction = d_view_dir;

            
            CUDA_CHECK(hipMemcpy(d_param, &params, sizeof(params), hipMemcpyHostToDevice));
            const OptixShaderBindingTable &sbt_ray_march = rtx_dataholder->sbt_ray_march;
            std::cout << "Launching Ray Tracer in Ray Marching Mode (" << width*height << " rays)\n";
            OPTIX_CHECK(optixLaunch(rtx_dataholder->pipeline_ray_march, inference_stream,
                                    reinterpret_cast<hipDeviceptr_t>(d_param),
                                    sizeof(Params), &sbt_ray_march, width, height, 1));
            CUDA_CHECK(hipStreamSynchronize(inference_stream));

            // CUDA Launch Sampling Kernel given entry and exit points from this perspective
            d_start_points = params.start_points;
            d_end_points = params.end_points;
            d_num_hits = params.num_hits;

            // print_intersections<<<1,1>>>(d_start_points, d_end_points, d_num_hits, 3 * grid_resolution);
            // CUDA_CHECK(hipDeviceSynchronize());

            std::cout << "Launching Sampling Kernel \n";
            //each point stores a location xyz and a viewing direction phi and psi
            
            int num_points;
            int samples_per_intersect = 32;
            // std::cout << "Print Num Hits \n";
            // print_int_arr<<<1,1>>>(d_num_hits, width, height);
            // CUDA_CHECK(hipDeviceSynchronize());

            // std::cout << "Print Viewdirs \n";
            // print_float2_arr<<<1,1>>>(d_view_dir, width, height);
            // CUDA_CHECK(hipDeviceSynchronize());

            thrust::device_ptr<int> dev_ptr_num_hits = thrust::device_pointer_cast(d_num_hits);
            num_points = thrust::reduce(dev_ptr_num_hits, dev_ptr_num_hits + width * height);
            thrust::device_vector<int> d_hit_indsV(width * height);
            // exclusive scan on dev_ptr_num_hits
            thrust::exclusive_scan(dev_ptr_num_hits, dev_ptr_num_hits + width * height, d_hit_indsV.begin());

            // convert dev_ptr_num_hits back to device int pointer
            d_num_hits = dev_ptr_num_hits.get();
            int *d_hit_inds = thrust::raw_pointer_cast(d_hit_indsV.data());
            // std::cout << "Print Num Hits post scan \n";
            // print_int_arr<<<1,1>>>(d_hit_inds, width, height);
            CUDA_CHECK(hipDeviceSynchronize());


            printf("num_hits_cu: %d\n", num_points);
            int num_sampled_points = samples_per_intersect * num_points;
            printf("sampled_points: %d\n", num_sampled_points);
            num_sampled_points = (num_sampled_points / batch_size) * batch_size + batch_size;
            printf("upsampled_points: %d\n", num_sampled_points);
            float* d_sampled_points;
            float* d_sampled_points_radiance;
            float* d_t_vals;
            unsigned int size_input = num_sampled_points * sizeof(float) * 5;
            unsigned int size_output = num_sampled_points * sizeof(float) * 4;
            printf("ALLOCATING %d bytes for samples (shouldn't be zero) \n", size_input);
            printf("ALLOCATING %d bytes for radiance (shouldn't be zero) \n", size_output);
            CUDA_CHECK(hipMalloc((void**)&d_sampled_points, size_input));
            CUDA_CHECK(hipMalloc((void**)&d_sampled_points_radiance,
                        size_output));
            CUDA_CHECK(hipMalloc((void**)&d_t_vals, sizeof(float) * num_sampled_points));

            launchSampler(
                d_start_points,
                d_end_points,
                d_view_dir,
                d_t_vals,
                d_sampled_points,
                width, height, grid_resolution,
                d_num_hits, d_hit_inds, 
                SAMPLING_REGULAR, inference_stream
            );
            CUDA_CHECK(hipDeviceSynchronize());
            // print_float5_arr<<<1,1>>>(d_sampled_points, num_sampled_points);
            // CUDA_CHECK(hipDeviceSynchronize());
            // print_float3_arr<<<1,1>>>(d_start_points, width * height * grid_resolution * 3);
            // CUDA_CHECK(hipDeviceSynchronize());
            // print_float3_arr<<<1,1>>>(d_end_points, width * height * grid_resolution * 3);
            // CUDA_CHECK(hipDeviceSynchronize());
            
            uint32_t padded_output_width = model.network->padded_output_width();
            tcnn::GPUMatrix<float> input_batch(n_input_dims, batch_size);
            tcnn::GPUMatrix<tcnn::network_precision_t> output_fwd(padded_output_width, batch_size);
            int num_iters = 0;
            for(int i = 0; i < num_sampled_points; i+=batch_size) {
                num_iters++;
                unsigned int offset = i * n_input_dims;
                CUDA_CHECK(hipMemcpy(
                    input_batch.data(),
                    d_sampled_points + offset,
                    batch_size * n_input_dims * sizeof(float),
                    hipMemcpyDeviceToDevice));
                
                auto ctx = model.network->forward(inference_stream, input_batch, &output_fwd, true);
                tcnn::GPUMatrix<tcnn::network_precision_t> output_slice = output_fwd.slice_rows(0, n_output_dims);
                // convert output_fwd to float
                int num_el = output_fwd.n_elements();
                int blockSize = 256;
                int numBlocks = (num_el + blockSize - 1) / blockSize;
                convertHalfToFloat<<<numBlocks,blockSize>>>(output_slice.data(), d_temp_out, batch_size * n_output_dims);
                CUDA_CHECK(hipDeviceSynchronize());
                CUDA_CHECK(hipMemcpy(
                    d_sampled_points_radiance + i * 4,
                    d_temp_out,
                    batch_size * n_output_dims * sizeof(float),
                    hipMemcpyDeviceToDevice));
            }
            //print radiance buffer values
            printf("Printing radiance buffer values\n");
            print_float4_arr<<<1,1>>>(d_sampled_points_radiance, num_sampled_points);
            CUDA_CHECK(hipDeviceSynchronize());

            // Launch Volume Rendering kernel
            printf("Launching Volume Rendering Kernel\n");
            

            launch_volrender_cuda(
                d_sampled_points,
                d_sampled_points_radiance,
                d_num_hits,
                d_hit_inds,
                d_t_vals,
                width,
                height,
                samples_per_intersect,
                d_pixels
            );
            
            
            // initialize host pixels
            float* h_pixels = new float[width * height * 3];
            // copy pixels to host
            CUDA_CHECK(hipMemcpy(
                h_pixels,
                d_pixels,
                width * height * sizeof(float) * 3,
                hipMemcpyDeviceToHost));
            CUDA_CHECK(hipDeviceSynchronize());
            
            // save pixels to png file with stb
            stbi_write_png("output.png", width, height, 3, h_pixels, width);
            size_t bufferSize = width * height * channels;
            int blockSize = 256;
            int numBlocks = (bufferSize + blockSize - 1) / blockSize;
            floatToHalf<<<numBlocks, blockSize>>>(d_pixels, d_pixels_half, bufferSize);
            hipDeviceSynchronize();
            tcnn::GPUMatrix<tcnn::network_precision_t> predicted_image(d_pixels_half, width * height, channels);

            // tcnn compute loss and backpropagate
            tcnn::GPUMatrix<float> target_image(d_image, width * height, channels);
            tcnn::GPUMatrix<float> values(width * height, channels);
            tcnn::GPUMatrix<tcnn::network_precision_t> gradients(width * height, channels);
            model.loss->evaluate(1.0f, predicted_image, target_image, values, gradients);
            float image_loss = tcnn::reduce_sum(values.data(), values.n_elements(), inference_stream);
            std::cout << "Image Loss: " << image_loss << std::endl;

            
	    break;
        }
        break;
    }
    return 0;
}
