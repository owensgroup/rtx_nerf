#include "hip/hip_runtime.h"
#include <iostream>
#include <string>
#include <vector>
#include <cstdio>
#include <fstream>
#include <math.h>
#include "stdio.h"


#include "optix_function_table_definition.h"
#include "optix_stubs.h"
#include "optix.h"
#include "optix_types.h"

#define STB_IMAGE_IMPLEMENTATION
#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "stb_image.h"
#include "stb_image_write.h"
#include "tiny-cuda-nn/common.h"
#include "tiny-cuda-nn/gpu_matrix.h"
#include <json/json.h>
#include "rtx/include/params.h"
#include "rtx/include/rtxFunctions.h"

#include "data_loader.h"

// Configure the model
nlohmann::json config = {
	{"loss", {
		{"otype", "L2"}
	}},
	{"optimizer", {
		{"otype", "Adam"},
		{"learning_rate", 1e-3},
	}},
	{"encoding", {
		{"otype", "HashGrid"},
		{"n_levels", 16},
		{"n_features_per_level", 2},
		{"log2_hashmap_size", 19},
		{"base_resolution", 16},
		{"per_level_scale", 2.0},
	}},
	{"network", {
		{"otype", "FullyFusedMLP"},
		{"activation", "ReLU"},
		{"output_activation", "None"},
		{"n_neurons", 64},
		{"n_hidden_layers", 2},
	}},
};
__global__ void printFloats(float* gpuPointer, int size)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    if (tid < size)
    {
        printf("Value at index %d: %f\n", tid, gpuPointer[tid]);
    }
}
__global__ void print_batch(float* batch, int batch_size, int image_size) {
    //printf("HELLO???\n");
    // int tid = blockIdx.x * blockDim.x + threadIdx.x;
    // if (tid < batch_size) {
    //     printf("Batch %d\n", tid);
    //     for (int i = 0; i < image_size; ++i) {
    //         printf("%f ", batch[tid * image_size + i]);
    //     }
    //     printf("\n");
    // }
}
// Creates a grid of Axis-aligned bounding boxes with specified resolution
// Bounding box coordinates are specified in normalized coordinates from -1 to 1
// TODO: make this a CUDA kernel
std::vector<OptixAabb> make_grid(int resolution) {
    std::vector<OptixAabb> grid;
    float box_length = 2.0f/ (float)resolution;
    for(int x = 0; x < resolution; x++) {
        for(int y = 0; y < resolution; y++) {
            for(int z = 0; z < resolution; z++) {
                OptixAabb aabb;
                aabb.minX = -1.0f + (float)x * box_length;
                aabb.maxX = -1.0f + x * box_length + box_length;
                aabb.minY = -1.0f + y * box_length;
                aabb.maxY = -1.0f + y * box_length + box_length;
                aabb.minZ = -1.0f + z * box_length;
                aabb.maxZ = -1.0f + z * box_length + box_length;
                grid.push_back(aabb);
                //std::printf("aabb (%.2f %.2f %.2f) (%.2f %.2f %.2f)\n",
                //        aabb.minX, aabb.minY, aabb.minZ, aabb.maxX, aabb.maxY, aabb.maxZ);
            }
        }
    }
    return grid;
}
//auto model = tcnn::create_from_config(n_input_dims, n_output_dims, config);

#define EPOCHS 10
#define BATCH_SIZE tcnn::batch_size_granularity
#define DATASET_SIZE 1000

RTXDataHolder *rtx_dataholder;

__global__ void print_intersections(float3* start, float3* end, int* num_hits, int num_prim) {
    printf("Intersections\n");
    for (int i = 0; i < 10; ++i) {
        printf("ray (%i): %i hits\n", i, num_hits[i]); // origin = (%.2f, %.2f, %.2f)\n  ",
        for (int j = 0; j < num_hits[i]; ++j) {
            float3 s = start[i*num_prim + j];
            float3 e = end[i*num_prim + j];
            printf("   (%.2f %.2f %.2f) (%.2f %.2f %.2f)\n", s.x, s.y, s.z, e.x, e.y, e.z);
        }
    }
}



int main() {
    // load data from files
    // TODO: take images and poses from json and load into DataLoader
    int num_epochs = EPOCHS;
    std::cout << "---------------------- Loading Data ----------------------\n";
    // Loads the Training, validation, and test sets from the synthetic lego scene
    std::vector<ImageDataset> datasets = load_data(SceneType::SYNTHETIC, SyntheticName::LEGO);
    auto train_set = datasets[0];
    unsigned int width = train_set.image_width;
    unsigned int height = train_set.image_height;
    unsigned int channels = train_set.image_channels;
    float training_focal = train_set.focal;
    float aspect_ratio = (float)width / (float)height;
    float focal_length = 1.0f / tan(0.5f * training_focal);
    size_t image_size = width * height * channels;
    // get training dataset from datasets
    std::vector<float*> training_images = datasets[0].images;
    std::vector<float*> training_poses = datasets[0].poses;
    std::cout << "---------------------- Data Loaded ----------------------\n\n\n";
    // Initialize our Optix Program Groups and Pipeline
    // We also build our initial dense acceleration structure of AABBs

    std::cout << "---------------------- Initializing Optix ----------------------\n";
    hipStream_t stream;
    CUDA_CHECK(hipStreamCreate(&stream));
    std::string ptx_filename = BUILD_DIR "bin/ptx/optixPrograms.ptx";

    rtx_dataholder = new RTXDataHolder();
    std::cout << "Initializing Context \n";
    rtx_dataholder->initContext();
    std::cout << "Reading PTX file and creating modules \n";
    rtx_dataholder->createModule(ptx_filename);
    std::cout << "Creating Optix Program Groups \n";
    rtx_dataholder->createProgramGroups();
    std::cout << "Linking Pipeline \n";
    rtx_dataholder->linkPipeline(false);
    std::cout << "Building Shader Binding Table (SBT) \n";
    rtx_dataholder->buildSBT();
    
    // Build our initial dense acceleration structure
    int grid_resolution = 8;
    std::cout << "Building Acceleration Structure \n";
    std::vector<OptixAabb> grid = make_grid(grid_resolution);
    int num_primitives = grid.size();
    
    OptixAabb* d_aabb = rtx_dataholder->initAccelerationStructure(grid);
    std::cout << "Done Building Acceleration Structure \n";
    std::cout << "---------------------- Done Initializing Optix ----------------------\n\n\n";

    std::cout << "Allocating Buffers on GPU" << std::endl;
    float* d_image, *d_look_at;
    CUDA_CHECK(hipMalloc((void **)&d_image, image_size * sizeof(float)));
    CUDA_CHECK(hipMalloc((void **)&d_look_at, 16 * sizeof(float)));
    std::cout << "Image Buffers Allocated on GPU" << std::endl;
    // Allocate buffers to hold outputs from ray intersection tests
    // start and end points are equal to # of AABBs in AS per ray [width * height * num_primitives]
    float3 *d_start_points;
    float3 *d_end_points;
    int *d_num_hits;
            
    CUDA_CHECK(hipMalloc((void **)&d_start_points, width * height * num_primitives * sizeof(float3)));
    CUDA_CHECK(hipMalloc((void **)&d_end_points, width * height * num_primitives * sizeof(float3)));
    CUDA_CHECK(hipMalloc((void **)&d_num_hits, width * height * sizeof(int)));
    std::cout << "Ray Intersection Buffers Allocated on GPU" << std::endl;

    Params *d_param;
    CUDA_CHECK(hipMalloc((void **)&d_param, sizeof(Params)));
    std::cout << "Params Buffer Allocated on GPU" << std::endl;


    // We train our neural network for a specific amount of epochs
    for (int j = 0; j < num_epochs; ++j) {
        std::printf("Started training loop epoch %d\n", j);
        
        // Loop through each set of images and poses in our training dataset
        for(int i = 0; i < training_images.size(); i++) {
            float* image = training_images[i];
            float* look_at = training_poses[i];

            // transfer image and look_at to GPU
            CUDA_CHECK(hipMemcpyAsync(d_image, image, image_size * sizeof(float), hipMemcpyHostToDevice, stream));
            CUDA_CHECK(hipMemcpyAsync(d_look_at, look_at, 16 * sizeof(float), hipMemcpyHostToDevice, stream));

            // Memset ray intersection buffers
            CUDA_CHECK(hipMemset(d_start_points, -2, width * height * num_primitives * sizeof(float3)));
            CUDA_CHECK(hipMemset(d_end_points, -2, width * height * num_primitives * sizeof(float3)));
            CUDA_CHECK(hipMemset(d_num_hits, 0, width * height * sizeof(int)));

            // Algorithmic parameters and data pointers used in GPU program
            Params params;
            // params.transform_matrix = transform_matrix;
            float d =  2.0f / grid_resolution;
            params.delta = make_float3(d, d, d);
            params.min_point = make_float3(-1, -1, -1);
            params.max_point = make_float3(1, 1, 1);
            params.width = width;
            params.height = height;
            params.focal_length = focal_length;
            params.aspect_ratio = aspect_ratio;
            params.handle = rtx_dataholder->gas_handle;
            params.aabb = d_aabb;
            params.start_points = d_start_points;
            params.end_points = d_end_points;
            params.num_hits = d_num_hits;
            params.num_primitives = num_primitives;

            
            CUDA_CHECK(hipMemcpy(d_param, &params, sizeof(params), hipMemcpyHostToDevice));
            const OptixShaderBindingTable &sbt_ray_march = rtx_dataholder->sbt_ray_march;
            std::cout << "Launching Ray Tracer in Ray Marching Mode \n";
            OPTIX_CHECK(optixLaunch(rtx_dataholder->pipeline_ray_march, stream,
                                    reinterpret_cast<hipDeviceptr_t>(d_param),
                                    sizeof(Params), &sbt_ray_march, width, height, 1));
            CUDA_CHECK(hipStreamSynchronize(stream));

            // CUDA Launch Sampling Kernel given entry and exit points from this perspective
            d_start_points = params.start_points;
            d_end_points = params.end_points;
            d_num_hits = params.num_hits;

            print_intersections<<<1,1>>>(d_start_points, d_end_points, d_num_hits, num_primitives);
            CUDA_CHECK(hipDeviceSynchronize());

            std::cout << "Launching Sampling Kernel \n";
            // tcnn inference on point buffer from sampling kernels
            
            // Optix Launch Volume Rendering kernel

            // tcnn compute loss and backpropagate

	    break;
        }
        break;
    }
    return 0;
}
