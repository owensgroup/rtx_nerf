#include "hip/hip_runtime.h"
#include "sampler.h"
#include <thrust/random.h>
#define BLOCK_SIZE 32
// Samples are returned from 0.0 to 1.0, where 0.0 is the same as start_points[0] and
// 1.0 is the same as the last end_point



// each thread computes the samples for one ray
// this should be launched with one thread per ray in an image
// each thread will read from num_hits
// each thread will compute samples for num_hits points
// start_points and end_points are the same size [width, height, grid_res * 3]
__global__ void generate_samples(
    float3* start_points,
    float3* end_points,
    float2* view_dirs,
    int batch_size,
    int grid_res,
    int* num_hits,
    int* indices,
    SAMPLING_TYPE sample_type,
    float* samples,
    float* t_vals,
    thrust::minstd_rand rng) 
{
    // Get index for this ray
    int x = threadIdx.x + blockDim.x * blockIdx.x;
    
    
    // get the viewing direction for this ray
    int start_index = indices[x];
    int n_hits = num_hits[x];
    float2 view_dir = view_dirs[x];
    float theta = view_dir.x;
    float phi = view_dir.y;
    if(x < batch_size) {
        for(int j = 0; j < n_hits; j++) {
            // grab the start and end points for this segment
            // start and end points have size [width, height, grid_res * 3]
            // find the start and end points for this thread
            float3 origin = start_points[start_index + j];
            float3 finish = end_points[start_index + j];
            float3 direction;
            direction.x = finish.x - origin.x;
            direction.y = finish.y - origin.y;
            direction.z = finish.z - origin.z;
            
            float t_initial = 0.0f;
            float t_final = 1.0f / NUM_SAMPLES_PER_SEGMENT;

            #pragma unroll
            for (int i = 0; i < NUM_SAMPLES_PER_SEGMENT; i++) {
                if (sample_type == SAMPLING_REGULAR) {
                    float t = t_initial;
                    float3 sample = origin;
                    sample.x = t * direction.x + origin.x;
                    sample.y = t * direction.y + origin.y;
                    sample.z = t * direction.z + origin.z;
                    samples[((start_index + j) * NUM_SAMPLES_PER_SEGMENT + i) * 5] = sample.x;
                    samples[((start_index + j) * NUM_SAMPLES_PER_SEGMENT + i) * 5 + 1] = sample.y;
                    samples[((start_index + j) * NUM_SAMPLES_PER_SEGMENT + i) * 5 + 2] = sample.z;
                    samples[((start_index + j) * NUM_SAMPLES_PER_SEGMENT + i) * 5 + 3] = theta;
                    samples[((start_index + j) * NUM_SAMPLES_PER_SEGMENT + i) * 5 + 4] = phi;
                    t_initial += 1.0f / NUM_SAMPLES_PER_SEGMENT;
                    t_vals[((start_index + j) * NUM_SAMPLES_PER_SEGMENT + i)] = t_initial;
                } 
                else if (sample_type == SAMPLING_UNIFORM) {
                    thrust::uniform_real_distribution<float> dist(0,1);
                    float t = dist(rng);
                    t_vals[((start_index + j) * NUM_SAMPLES_PER_SEGMENT + i)] = t_initial;
                    float3 sample = origin;
                    sample.x = t * direction.x + origin.x;
                    sample.y = t * direction.y + origin.y;
                    sample.z = t * direction.z + origin.z;
                    
                    samples[((start_index + j) * NUM_SAMPLES_PER_SEGMENT + i) * 5] = sample.x;
                    samples[((start_index + j) * NUM_SAMPLES_PER_SEGMENT + i) * 5 + 1] = sample.y;
                    samples[((start_index + j) * NUM_SAMPLES_PER_SEGMENT + i) * 5 + 2] = sample.z;
                    samples[((start_index + j) * NUM_SAMPLES_PER_SEGMENT + i) * 5 + 3] = theta;
                    samples[((start_index + j) * NUM_SAMPLES_PER_SEGMENT + i) * 5 + 4] = phi;
                } else if (sample_type == SAMPLING_STRATIFIED_JITTERING) {
                    thrust::uniform_real_distribution<float> dist(t_initial, t_final);
                    float t = dist(rng);

                    float3 sample = origin;
                    sample.x = t * direction.x + origin.x;
                    sample.y = t * direction.y + origin.y;
                    sample.z = t * direction.z + origin.z;

                    samples[((start_index + j) * NUM_SAMPLES_PER_SEGMENT + i) * 5] = sample.x;
                    samples[((start_index + j) * NUM_SAMPLES_PER_SEGMENT + i) * 5 + 1] = sample.y;
                    samples[((start_index + j) * NUM_SAMPLES_PER_SEGMENT + i) * 5 + 2] = sample.z;
                    samples[((start_index + j) * NUM_SAMPLES_PER_SEGMENT + i) * 5 + 3] = theta;
                    samples[((start_index + j) * NUM_SAMPLES_PER_SEGMENT + i) * 5 + 4] = phi;            
                    t_vals[((start_index + j) * NUM_SAMPLES_PER_SEGMENT + i)] = t_initial;
                    t_initial = t_final;
                    t_final += 1.0f / NUM_SAMPLES_PER_SEGMENT;
                }
            }
        }
    }
}

void launchSampler(
    float3* d_start_points,
    float3* d_end_points,
    float2* d_view_dirs,
    float* d_t_vals,
    float* d_sampled_points,
    int batch_size,
    int grid_res,
    int* d_num_hits,
    int* d_indices,
    SAMPLING_TYPE sample_type, 
    hipStream_t& stream) {
        thrust::minstd_rand rng;
        dim3 block(1024);
        dim3 grid((batch_size + block.x - 1) / block.x);
        generate_samples<<<grid, block,0,stream>>>(
            d_start_points,
            d_end_points,
            d_view_dirs,
            batch_size, grid_res,
            d_num_hits, d_indices,
            sample_type,
            d_sampled_points,
            d_t_vals,
            rng
        );
    }