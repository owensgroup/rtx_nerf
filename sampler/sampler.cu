#include "hip/hip_runtime.h"
#include "sampler.h"
#include <thrust/random.h>
#define BLOCK_SIZE 32
// Samples are returned from 0.0 to 1.0, where 0.0 is the same as start_points[0] and
// 1.0 is the same as the last end_point



// each thread computes the samples for one ray
// this should be launched with one thread per ray in an image
// each thread will read from num_hits
// each thread will compute samples for num_hits points
// start_points and end_points are the same size [width, height, grid_res * 3]

__global__ void generate_samples(
    float3* start_points,
    float3* end_points,
    float2* view_dirs,
    int width,
    int height,
    int grid_res,
    int* num_hits,
    int* indices,
    SAMPLING_TYPE sample_type,
    float* samples,
    thrust::minstd_rand rng) 
{
    // Get index for this ray
    int x = threadIdx.x + blockDim.x * blockIdx.x;
    int y = threadIdx.y + blockDim.y * blockIdx.y;
    // get the viewing direction for this ray
    int start_index = indices[y * width + x];
    float2 view_dir = view_dirs[y * width + x];
    float theta = view_dir.x;
    float phi = view_dir.y;

    if(x < width && y < height) {
        for(int j = 0; j < num_hits[y * width + x]; j++) {
            // grab the start and end points for this segment
            // start and end points have size [width, height, grid_res * 3]
            // find the start and end points for this thread
            int idx = (y * width * grid_res * 3) + (x * grid_res * 3) + j;
            float3 origin = start_points[idx];
            float3 finish = end_points[idx];
            float3 direction;
            direction.x = finish.x - origin.x;
            direction.y = finish.y - origin.y;
            direction.z = finish.z - origin.z;
            
            float t_initial = 0.0f;
            float t_final = 1.0f / NUM_SAMPLES_PER_SEGMENT;

            #pragma unroll
            for (int i = 0; i < NUM_SAMPLES_PER_SEGMENT; i++) {
                if (sample_type == SAMPLING_REGULAR) {
                    float t = t_initial;
                    float3 sample = origin;
                    sample.x = t * direction.x + origin.x;
                    sample.y = t * direction.y + origin.y;
                    sample.z = t * direction.z + origin.z;
                    printf("Thread %d, Writing to sample index %d\n", y*width+x, ((start_index + j) * NUM_SAMPLES_PER_SEGMENT + i) * 5);
                    samples[((start_index + j) * NUM_SAMPLES_PER_SEGMENT + i) * 5] = sample.x;
                    samples[((start_index + j) * NUM_SAMPLES_PER_SEGMENT + i) * 5 + 1] = sample.y;
                    samples[((start_index + j) * NUM_SAMPLES_PER_SEGMENT + i) * 5 + 2] = sample.z;
                    samples[((start_index + j) * NUM_SAMPLES_PER_SEGMENT + i) * 5 + 3] = theta;
                    samples[((start_index + j) * NUM_SAMPLES_PER_SEGMENT + i) * 5 + 4] = phi;
                    t_initial += 1.0f / NUM_SAMPLES_PER_SEGMENT;
                } else if (sample_type == SAMPLING_UNIFORM) {
                    thrust::uniform_real_distribution<float> dist(0,1);
                    float t = dist(rng);

                    float3 sample = origin;
                    sample.x = t * direction.x + origin.x;
                    sample.y = t * direction.y + origin.y;
                    sample.z = t * direction.z + origin.z;
                    
                    samples[((start_index + j) * NUM_SAMPLES_PER_SEGMENT + i) * 5] = sample.x;
                    samples[((start_index + j) * NUM_SAMPLES_PER_SEGMENT + i) * 5 + 1] = sample.y;
                    samples[((start_index + j) * NUM_SAMPLES_PER_SEGMENT + i) * 5 + 2] = sample.z;
                    samples[((start_index + j) * NUM_SAMPLES_PER_SEGMENT + i) * 5 + 3] = theta;
                    samples[((start_index + j) * NUM_SAMPLES_PER_SEGMENT + i) * 5 + 4] = phi;
                } else if (sample_type == SAMPLING_STRATIFIED_JITTERING) {
                    thrust::uniform_real_distribution<float> dist(t_initial, t_final);
                    float t = dist(rng);

                    float3 sample = origin;
                    sample.x = t * direction.x + origin.x;
                    sample.y = t * direction.y + origin.y;
                    sample.z = t * direction.z + origin.z;

                    samples[((start_index + j) * NUM_SAMPLES_PER_SEGMENT + i) * 5] = sample.x;
                    samples[((start_index + j) * NUM_SAMPLES_PER_SEGMENT + i) * 5 + 1] = sample.y;
                    samples[((start_index + j) * NUM_SAMPLES_PER_SEGMENT + i) * 5 + 2] = sample.z;
                    samples[((start_index + j) * NUM_SAMPLES_PER_SEGMENT + i) * 5 + 3] = theta;
                    samples[((start_index + j) * NUM_SAMPLES_PER_SEGMENT + i) * 5 + 4] = phi;            

                    t_initial = t_final;
                    t_final += 1.0f / NUM_SAMPLES_PER_SEGMENT;
                }
            }
    
        }
    }
}

void launchSampler(
    float3* d_start_points,
    float3* d_end_points,
    float2* d_view_dirs,
    float* d_sampled_points,
    unsigned int width, 
    unsigned int height,
    int grid_res,
    int* d_num_hits,
    int* d_indices,
    SAMPLING_TYPE sample_type, 
    hipStream_t& stream) {
        thrust::minstd_rand rng;
        dim3 block(BLOCK_SIZE, BLOCK_SIZE);
        dim3 grid((width + block.x - 1) / block.x, (height + block.y - 1) / block.y);
        generate_samples<<<grid, block,0,stream>>>(
            d_start_points,
            d_end_points,
            d_view_dirs,
            width,
            height,
            grid_res,
            d_num_hits,
            d_indices,
            sample_type,
            d_sampled_points,
            rng
        );
    }