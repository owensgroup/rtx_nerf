#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2020, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

 //include printf
#include <stdio.h>
#include <optix.h>

#include "params.h"
#include <optix.h>


extern "C" static __constant__ Params params;

extern "C" __global__ void __raygen__ray_march() {
  const uint3 launch_index = optixGetLaunchIndex();
  const float3 &delta = params.delta;
  const float3 &min_point = params.min_point;
  const float3 &max_point = params.max_point;

  // generate ray origin
  // TODO: Apply transform matrix to ray origin
  // look_at is the 4x4 transform matrix
  float* look_at = params.look_at;

  float xo = min_point.x + delta.x * launch_index.x + (delta.x / 2);
  float yo = min_point.y + delta.y * launch_index.y + (delta.y / 2);
  float zo = min_point.z;
   
  float3 ray_origin = make_float3(xo, yo, zo);

  float3 ray_direction = make_float3(0.0, 0.0, 1.0);

  float tmin = 0.0f;
  float tmax = (max_point.z - min_point.z) + 100.0;
  float ray_time = 0.0f;
  // Visibility mask is used to mask out objects from rays
  // for each part of the scene, a mask is assigned and when
  // the ray intersects a bitwise and is performed 
  OptixVisibilityMask visibilityMask = 255;
  unsigned int rayFlags = OPTIX_RAY_FLAG_NONE;
  unsigned int SBToffset = 0;
  unsigned int SBTstride = 0;
  unsigned int missSBTIndex = 0;
  unsigned int payload = 0;
  optixTrace(params.handle, ray_origin, ray_direction, tmin, tmax, ray_time,
             visibilityMask, rayFlags, SBToffset, SBTstride, missSBTIndex,
             payload);

  unsigned int idx = launch_index.x + launch_index.y * params.width;
  params.start_points[idx].x = __uint_as_float(payload);
}

extern "C" __global__ void __anyhit__ray_march() {
  // For every intersection, we update a scalar value
  float val = __uint_as_float(optixGetPayload_0());
  val += 0.2; // can be the scalar value associated with a triangle.
  optixSetPayload_0(__float_as_uint(val));
  optixIgnoreIntersection();
}

// extern "C" __global__ void __miss__ray_march() {}
extern "C" __global__ void __closesthit__ray_march() {
  // For every closest hit, grab the entry and exit point for the ray
  const uint3 launch_index = optixGetLaunchIndex();
  float t_min = optixGetRayTmin();
  float t_max = optixGetRayTmax(); // t_max returns smallest reported hitT

  // compute the ray entry point from t_min
  float3 ray_direction = optixGetWorldRayDirection();
  float3 ray_origin = optixGetWorldRayOrigin();
  float s_x = ray_origin.x + t_min * ray_direction.x;
  float s_y = ray_origin.y + t_min * ray_direction.y;
  float s_z = ray_origin.z + t_min * ray_direction.z;
  float3 start = make_float3(s_x, s_y, s_z);

  // compute the ray exit point from t_max
  float e_x = ray_origin.x + t_max * ray_direction.x;
  float e_y = ray_origin.y + t_max * ray_direction.y;
  float e_z = ray_origin.z + t_max * ray_direction.z;
  float3 end = make_float3(e_x, e_y, e_z);

  // get the optixAabb that we currenty intersected with
  uint primitiveIndex = optixGetPrimitiveIndex();

  // store the entry and exit points of this ray in this AABB in param buffers
  // entry_points is an array with dimension [H, W, numPrimitives]
  // exit_points is an array with dimension [H, W, numPrimitives]
  unsigned int idx = primitiveIndex + launch_index.x * params.num_primitives +
                     launch_index.y * params.num_primitives * params.width;
  params.start_points[idx] = start;
  params.end_points[idx] = end;

  // update the number of intersections for this ray
  unsigned int idx2 = launch_index.x + launch_index.y * params.width;
  params.num_hits[idx2] += 1;

  // relaunch a ray from the exit point of the current AABB
  // this ray will be used to find the next AABB that the ray intersects with
  // the ray will be launched in the same direction as the original ray
  OptixVisibilityMask visibilityMask = 255;
  unsigned int rayFlags = OPTIX_RAY_FLAG_NONE;
  unsigned int SBToffset = 0;
  unsigned int SBTstride = 0;
  unsigned int missSBTIndex = 0;
  unsigned int payload = 0;
  optixTrace(params.handle, end, ray_direction, 0, 100, 0,
             visibilityMask, rayFlags, SBToffset, SBTstride, missSBTIndex,
             payload);
}

extern "C" __global__ void __raygen__ray_sample() {
  const uint3 launch_index = optixGetLaunchIndex();
  const float3 &delta = params.delta;
  const float3 &min_point = params.min_point;
  //const float* transform_matrix = params.transform_matrix;
  float xo = min_point.x + delta.x * launch_index.x + (delta.x / 2);
  float yo = min_point.y + delta.y * launch_index.y + (delta.y / 2);
  float zo = -1 * (min_point.z + delta.z * launch_index.z);
  float3 ray_origin = make_float3(xo, yo, zo);

  float3 ray_direction = make_float3(0.0, 0.0, 1.0);

  float tmin = 0.0f;
  float tmax = delta.z + 1;
  float ray_time = 0.0f;
  OptixVisibilityMask visibilityMask = 255;
  unsigned int rayFlags = OPTIX_RAY_FLAG_DISABLE_ANYHIT;
  unsigned int SBToffset = 0;
  unsigned int SBTstride = 0;
  unsigned int missSBTIndex = 0;

  // Save exit points of each AABB in optixPayload (setoptixpayload)
  // relaunch optix trace replace ray origins with exit points from optix payload
  // may need another payload to know when to stop launching rays
  
  // tmax should be 
  optixTrace(params.handle, ray_origin, ray_direction, tmin, tmax, ray_time,
             visibilityMask, rayFlags, SBToffset, SBTstride, missSBTIndex);
}

// extern "C" __global__ void __anyhit__ray_sample() {}
// extern "C" __global__ void __miss__ray_sample() {}

extern "C" __global__ void __closesthit__ray_sample() {
  const uint3 launch_index = optixGetLaunchIndex();
  // For every closest hit, we update a scalar value in Global memory
  unsigned int idx = launch_index.x + launch_index.y * params.width;
  //float *output = params.output;

  // get t max value of ray for a hit
  // float t_current = optixGetRayTmax();
  // float3 ray_direction = optixGetWorldRayDirection();
  // float3 ray_origin = optixGetWorldRayOrigin();
  // float3 hit_point = ray_origin + t_current * ray_direction;

  // from the hitpoint march through till the exit point of the current AABB
  // store the sampled points in a buffer in Params
  // determine start point and end point
  // compute the end point given bounding box coordinates
  
  // compute the start and end points
  // store them in a buffer in Params
  // get the optixAabb that we currenty intersected with
  // use the bounding box coordinates in order to compute the end point
  // store the start and end points of this ray in this AABB in param buffers
  // uint primitiveIndex = optixGetPrimitiveIndex();
  // OptixAabb aabb = Params.aabb[primitiveIndex];
  // float3 min_point = make_float3(aabb.minX, aabb.minY, aabb.minZ);
  // float3 max_point = make_float3(aabb.maxX, aabb.maxY, aabb.maxZ);

  // compute the end point
  // this should store sampled points in a buffer in Params
  // atomicAdd(output + idx,
  //           0.2f); // can be the scalar value associated with a triangle.
}
